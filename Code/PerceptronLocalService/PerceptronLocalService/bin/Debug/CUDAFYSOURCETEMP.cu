
#include <hip/hip_runtime.h>

// PerceptronLocalService.Engine.PstGeneratorGpu
extern "C" __global__  void PstGenerationKernel( double* peaks, int peaksLen0,  int* peakListCountArray, int peakListCountArrayLen0,  double* aminoAcidMasses, int aminoAcidMassesLen0,  double* hopTolerance, int hopToleranceLen0,  unsigned short* pstMatchMatrix, int pstMatchMatrixLen0, int pstMatchMatrixLen1,  double* errors, int errorsLen0, int errorsLen1,  double* AveragePstIntensity, int AveragePstIntensityLen0, int AveragePstIntensityLen1);
// PerceptronLocalService.Engine.PstGeneratorGpu
extern "C" __global__  void GpuGeneratedAverageIntensities( double* IntensityPeakDataDevice, int IntensityPeakDataDeviceLen0,  int* intensitypeakListLengthDevice, int intensitypeakListLengthDeviceLen0,  double* GpuIntensityPeakMatrix, int GpuIntensityPeakMatrixLen0, int GpuIntensityPeakMatrixLen1);

// PerceptronLocalService.Engine.PstGeneratorGpu
extern "C" __global__  void PstGenerationKernel( double* peaks, int peaksLen0,  int* peakListCountArray, int peakListCountArrayLen0,  double* aminoAcidMasses, int aminoAcidMassesLen0,  double* hopTolerance, int hopToleranceLen0,  unsigned short* pstMatchMatrix, int pstMatchMatrixLen0, int pstMatchMatrixLen1,  double* errors, int errorsLen0, int errorsLen1,  double* AveragePstIntensity, int AveragePstIntensityLen0, int AveragePstIntensityLen1)
{
	int num = peakListCountArray[(0)];
	int num2 = threadIdx.x + blockIdx.x * blockDim.x;
	int num3 = num2 % num;
	int num4 = num2 / (num * 26);
	int num5 = num2 / num % 26;
	if (num4 < num && num4 > num3)
	{
		double num6 = peaks[(num4)] - peaks[(num3)];
		if (num6 < 0.0)
		{
			num6 *= -1.0;
		}
		double num7 = num6 - aminoAcidMasses[(num5)];
		if (num7 < 0.0)
		{
			num7 *= -1.0;
		}
		if (num7 < hopTolerance[(0)])
		{
			pstMatchMatrix[(num3) * pstMatchMatrixLen1 + ( num4)] = (unsigned short)num5;
			errors[(num3) * errorsLen1 + ( num4)] = num7;
		}
	}
}
// PerceptronLocalService.Engine.PstGeneratorGpu
extern "C" __global__  void GpuGeneratedAverageIntensities( double* IntensityPeakDataDevice, int IntensityPeakDataDeviceLen0,  int* intensitypeakListLengthDevice, int intensitypeakListLengthDeviceLen0,  double* GpuIntensityPeakMatrix, int GpuIntensityPeakMatrixLen0, int GpuIntensityPeakMatrixLen1)
{
	int num = threadIdx.x + blockIdx.x * blockDim.x;
	int num2 = num % intensitypeakListLengthDevice[(0)];
	int num3 = num2 + num / intensitypeakListLengthDevice[(0)];
	if (num3 < intensitypeakListLengthDevice[(0)] && num3 > num2)
	{
		double num4 = (IntensityPeakDataDevice[(num2)] + IntensityPeakDataDevice[(num3)]) / 2.0;
		GpuIntensityPeakMatrix[(num2) * GpuIntensityPeakMatrixLen1 + ( num3)] = num4;
	}
}
