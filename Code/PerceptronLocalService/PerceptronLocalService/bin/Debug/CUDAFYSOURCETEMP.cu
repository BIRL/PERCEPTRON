
#include <hip/hip_runtime.h>

// PerceptronLocalService.Engine.WholeProteinMassTunerGpu
extern "C" __global__  void DeviceGenerateFragmentPairMatrix( double* peaks, int peaksLen0,  int* len, int lenLen0,  double* output, int outputLen0, int outputLen1);

// PerceptronLocalService.Engine.WholeProteinMassTunerGpu
extern "C" __global__  void DeviceGenerateFragmentPairMatrix( double* peaks, int peaksLen0,  int* len, int lenLen0,  double* output, int outputLen0, int outputLen1)
{
	int num = threadIdx.x + blockIdx.x * blockDim.x;
	int num2 = num % len[(0)];
	int num3 = num2 + num / len[(0)];
	bool flag = num3 >= len[(0)];
	if (!flag)
	{
		double num4 = peaks[(num3)] + peaks[(num2)];
		output[(num2) * outputLen1 + ( num3)] = num4;
	}
}
