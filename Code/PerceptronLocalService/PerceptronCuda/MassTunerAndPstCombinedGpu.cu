#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hipblas.h>
#include <vector>
#include <assert.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include  <device_atomic_functions.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <string>
#include <iterator>
#include<algorithm>
#include <chrono>
#include <math.h>
#include "CudaHeaderFile.cuh"
#include "CudaHeaderForPst.cuh"

#define SIZE	1024 
using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#pragma pack(1)
typedef struct MassTunerAndPstCombinedStruct
{
	int PstTagLength;	
	double PstErrorScore;
	double PstFrequency;
	double MassTuner;
	char PstTags[8];
}masstunerandpstcombinedstruct;

void AccomodateIsoforms(vector< _PeptideSequenceTags> &Final_MultipleLengthPsts, ParametersToCpp Parameters)
{
	char ResidueForReplacement[] = { 'L', 'D', 'N', 'E', 'Q' };
	char newResidue;
	int CountOfFinalPsts = Final_MultipleLengthPsts.size();
	int g = strlen(ResidueForReplacement);
	for (int i = 0; i < Final_MultipleLengthPsts.size(); i++)
	{
		for (int j = 0; j < 5; j++)
		{
			char OldResidue = ResidueForReplacement[j];
			if (strchr(Final_MultipleLengthPsts[i].PstTag, OldResidue))
			{
				if (OldResidue == 'L')//Here I think Switch Case will be more better....!!!!
					newResidue = 'I';
				else if (OldResidue == 'D')
					newResidue = 'B';
				else if (OldResidue == 'N')
					newResidue = 'B';
				else if (OldResidue == 'E')
					newResidue = 'Z';
				else if (OldResidue == 'Q' && Parameters.HopThreshhold <= 1.5)
					newResidue = 'Z';
				else if (OldResidue == 'Q' && Parameters.HopThreshhold > 1.5)
					newResidue = 'K';

				string BeforeAccomodatePst = Final_MultipleLengthPsts[i].PstTag;
				_PeptideSequenceTags AccomodatedPstTag = Final_MultipleLengthPsts[i];
				for (int iter = 0; iter < Final_MultipleLengthPsts[i].PstTagLength; iter++)
				{
					if (BeforeAccomodatePst[iter] == OldResidue)
					{
						BeforeAccomodatePst[iter] = newResidue;
						strcpy(AccomodatedPstTag.PstTag, BeforeAccomodatePst.c_str());
						Final_MultipleLengthPsts.push_back(AccomodatedPstTag);
					}
				}
			}
		}
	}
	for (int i = 0; i < Final_MultipleLengthPsts.size() - 1; i++)
	{
		if (strcmp(Final_MultipleLengthPsts[i].PstTag, Final_MultipleLengthPsts[i + 1].PstTag) == 0)
		{
			Final_MultipleLengthPsts.erase(Final_MultipleLengthPsts.begin() + i + 1);
			i--;
		}
	}
}

void FindingUniquePSTs(vector< _PeptideSequenceTags> &Final_MultipleLengthPsts)
{
	for (int i = 0; i < Final_MultipleLengthPsts.size() - 1; i++)	// extracting the unique PSTs from the total PSTs obtained. If 2 PST tags are same, their error sums and intensities are compared
	{
		if (strcmp(Final_MultipleLengthPsts[i].PstTag, Final_MultipleLengthPsts[i + 1].PstTag) == 0)
		{
			if (Final_MultipleLengthPsts[i].ErrorSum < Final_MultipleLengthPsts[i + 1].ErrorSum)
			{
				Final_MultipleLengthPsts.erase(Final_MultipleLengthPsts.begin() + i + 1);
				i--;
			}
			else if (Final_MultipleLengthPsts[i].ErrorSum > Final_MultipleLengthPsts[i + 1].ErrorSum)
			{
				Final_MultipleLengthPsts.erase(Final_MultipleLengthPsts.begin() + i);
				i--;
			}
			else
			{
				if (Final_MultipleLengthPsts[i].PstFrequency <= Final_MultipleLengthPsts[i + 1].PstFrequency)
				{
					Final_MultipleLengthPsts.erase(Final_MultipleLengthPsts.begin() + i);
					i--;
				}
				else if (Final_MultipleLengthPsts[i].PstFrequency > Final_MultipleLengthPsts[i + 1].PstFrequency)
				{
					Final_MultipleLengthPsts.erase(Final_MultipleLengthPsts.begin() + i + 1);
					i--;
				}
			}
		}
	}
}

vector<_PeptideSequenceTags> CalculatingPeptideSequenceTags(thrust::host_vector<_DataForPsts> Host_SingleLengthPSTs, ParametersToCpp Parameters, int zN)
{
	thrust::device_vector<_DataForPsts> Final_SingleLengthPSTs;
	vector<_DataForPsts> lol;
	for (int i = 0; i < zN; i++)	// Populating the single length PSTs calculated while running mass tuner function
	{
		if (Host_SingleLengthPSTs[i].startIndexMass == 0)
			break;
		Final_SingleLengthPSTs.push_back(Host_SingleLengthPSTs[i]);
		lol.push_back(Host_SingleLengthPSTs[i]);
	}
	lol[0] = lol[0];
	_DataForPsts *Final_SingleLengthPSTs_ptr = thrust::raw_pointer_cast(Final_SingleLengthPSTs.data());	// Pointer pointing to single length PST vector that is to be sent to gpu

	thrust::device_vector<_PeptideSequenceTags> dev_MultipleLengthPst(1000);
	_PeptideSequenceTags *MultipleLengthPst_ptr = thrust::raw_pointer_cast(dev_MultipleLengthPst.data());	// multiple length PSTs pointer

	int numOfThr = Final_SingleLengthPSTs.size();	// each single length PST will be compared with the others in each thread
	int THREADSforPst = 256;	// optimal num of threads per block
	int BLOCKSforPst = numOfThr / THREADSforPst + 5;	// total threads we want to launch and the optimal threads will give us the num of blocks to be launched. +5 gist foe safety
	GeneratingMultipleLengthPsts << <BLOCKSforPst, THREADSforPst >> > (MultipleLengthPst_ptr, Final_SingleLengthPSTs_ptr, numOfThr);	// Here duplets of PSTs will be created in GPU
	thrust::host_vector<_PeptideSequenceTags> host_MultipleLengthPst = dev_MultipleLengthPst;	// Copying data back to CPU from GPU
	thrust::device_vector<_PeptideSequenceTags> dev_MultipleLengthPst2;

	for (int i = 0; i < host_MultipleLengthPst.size(); i++)	// Extracting the duplets and chopping the excess data so these duplets can be passed to gpu again to compute triplets
	{
		if (host_MultipleLengthPst[i].endIndex == 0)
			break;
		dev_MultipleLengthPst2.push_back(host_MultipleLengthPst[i]);
	}
	_PeptideSequenceTags *MultipleLengthPst_ptr2 = thrust::raw_pointer_cast(dev_MultipleLengthPst2.data());

	int size = 0;
	thrust::host_vector<_PeptideSequenceTags> host_MultipleLengthPst2;

	for (int num = 3; num <= Parameters.MaximumPstLength; num++)	// loop runs to calculate triplets, tetraplets, pentaplets and so on according to user defined parameters
	{
		numOfThr = dev_MultipleLengthPst2.size();
		int PstSize = num;
		GeneratingMultipleLengthPsts2 << <BLOCKSforPst, THREADSforPst >> > (MultipleLengthPst_ptr, MultipleLengthPst_ptr2, Final_SingleLengthPSTs_ptr, numOfThr, Final_SingleLengthPSTs.size(), PstSize);	// gpu kernel to calculate multiple length psts
		host_MultipleLengthPst = dev_MultipleLengthPst;

		size = size + dev_MultipleLengthPst2.size();
		dev_MultipleLengthPst2.clear();
		for (int i = size; i < host_MultipleLengthPst.size(); i++)	// Extracting the PSTs and chopping the excess data so these PSTs can be passed to gpu again to compute further longer length PSTs
		{
			if (host_MultipleLengthPst[i].endIndex == 0)
				break;
			dev_MultipleLengthPst2.push_back(host_MultipleLengthPst[i]);
		}
	}

	vector<_PeptideSequenceTags> Final_MultipleLengthPsts;
	for (int i = 0; i < host_MultipleLengthPst.size(); i++)	// Final multiple length PSTs being stored into vector according to user defined min and max PST length
	{
		if (host_MultipleLengthPst[i].endIndex == 0)
			break;
		if (host_MultipleLengthPst[i].PstTagLength >= Parameters.MinimumPstLength && host_MultipleLengthPst[i].PstTagLength <= Parameters.MaximumPstLength)
		{
			Final_MultipleLengthPsts.push_back(host_MultipleLengthPst[i]);
		}
	}

	std::sort(Final_MultipleLengthPsts.begin(), Final_MultipleLengthPsts.end(),	// sorting the data according to the PST tags
		[](const _PeptideSequenceTags &pst1, const _PeptideSequenceTags &pst2)
	{ return strcmp(pst1.PstTag, pst2.PstTag) < 0; });

	Final_MultipleLengthPsts[0] = Final_MultipleLengthPsts[0];	// DELME Just for checking values
	
	FindingUniquePSTs(Final_MultipleLengthPsts);	// Function call to extract unique PSTs
	
	Final_MultipleLengthPsts[0] = Final_MultipleLengthPsts[0];	// DELME Just for checking values

	AccomodateIsoforms(Final_MultipleLengthPsts, Parameters);	// Isoforms are being accomodated here and then their unique is taken

	Final_MultipleLengthPsts[0] = Final_MultipleLengthPsts[0];	// DELME Just for checking values
	return Final_MultipleLengthPsts;
}

extern "C" __declspec(dllexport) int __cdecl
wholeproteinmasstunerandpst(double PeakListMasses[], double PeakListIntensities[], int PeakListLength, ParametersToCpp Parameters, MassTunerAndPstCombinedStruct **_MassTunerAndPstCombinedStruct)
{	
	auto start = chrono::steady_clock::now();

	double WholeProteinMass = PeakListMasses[0];

	const int N = PeakListLength;
	const int zN = (floor(PeakListLength*PeakListLength) / 2) - (floor(PeakListLength / 2));
	double *dev_masses, *dev_intensities;

	double aminoAcidMassesList[21] = { 57.0214600000000, 71.0371100000000, 87.0320300000000, 97.0527600000000, 99.0684100000000, 101.047680000000, 103.009190000000, 113.084060000000, 114.042930000000, 115.026940000000, 128.058580000000, 128.094960000000, 129.042590000000, 131.040490000000, 137.058910000000, 147.068410000000, 156.101110000000, 163.063330000000, 168.964203000000, 186.079310000000, 255.158295000000 };
	char aminoAcidSymbolList[21] = { 'G', 'A', 'S', 'P', 'V', 'T', 'C', 'L', 'N', 'D', 'Q', 'K', 'E', 'M', 'H', 'F', 'R', 'Y', 'U', 'W', 'O' };
	
	double *dev_aminoAcidMassesList; char *dev_aminoAcidSymbolList;

	hipMalloc((void**)&dev_masses, sizeof(double) * N);
	hipMalloc((void**)&dev_intensities, sizeof(double) * N);
	hipMalloc((void**)&dev_aminoAcidMassesList, sizeof(double)*21);
	hipMalloc((void**)&dev_aminoAcidSymbolList, sizeof(char)*21);
	

	thrust::device_vector<double> DevicePeakListMassesSum(zN);
	double *DevicePeakListMassesSum_ptr = thrust::raw_pointer_cast(DevicePeakListMassesSum.data());
	thrust::device_vector<double> DevicePeakListAvgIntensities(zN);
	double *DevicePeakListAvgIntensities_ptr = thrust::raw_pointer_cast(DevicePeakListAvgIntensities.data());
	thrust::device_vector<_DataForPsts> DeviceSingleLengthPSTs(zN);
	_DataForPsts *SingleLengthPSTs_ptr = thrust::raw_pointer_cast(DeviceSingleLengthPSTs.data());

	hipMemcpy(dev_masses, PeakListMasses, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_intensities, PeakListIntensities, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_aminoAcidMassesList, aminoAcidMassesList, 21 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_aminoAcidSymbolList, aminoAcidSymbolList, 21 * sizeof(char), hipMemcpyHostToDevice);
	
	int THREADS = 256;
	int BLOCKS = (N/THREADS + 5);
	
	CalculatingTupleSumsAndSingleLengthPsts << < BLOCKS, THREADS >> > (dev_masses, dev_intensities, DevicePeakListMassesSum_ptr, DevicePeakListAvgIntensities_ptr, SingleLengthPSTs_ptr, dev_aminoAcidMassesList, dev_aminoAcidSymbolList, Parameters.MwTolerance, Parameters.NeutralLoss, Parameters.HopThreshhold, N, Parameters.Autotune, Parameters.DenovoAllow);
	

	thrust::host_vector<_DataForPsts> Host_SingleLengthPSTs = DeviceSingleLengthPSTs;
	
	double TunedMass = 0.0;
	if (Parameters.Autotune == 1)
	{
		thrust::host_vector<double> PeakListMassesSum = DevicePeakListMassesSum;
		thrust::host_vector<double> PeakListIntensitiesAverage = DevicePeakListAvgIntensities;
		thrust::host_vector<_ShortlistedMassSumsAndIntensities> shortlistedMassSumAndIntensities;

		for (int i = 0; i < zN; i++)
		{
			if (PeakListMasses[0] - Parameters.MwTolerance <= PeakListMassesSum[i] && PeakListMassesSum[i] <= PeakListMasses[0] + Parameters.MwTolerance)
			{
				_ShortlistedMassSumsAndIntensities data;
				data.massSum = PeakListMassesSum[i];
				data.AvgIntensity = PeakListIntensitiesAverage[i];
				shortlistedMassSumAndIntensities.push_back(data);
			}
		}

		if (shortlistedMassSumAndIntensities.size() == 0) {  // When short list (thrust vector) sum of masses and their intensities will be zero //Updated 20210305
			hipDeviceSynchronize();
			return TunedMass = 0.0;
		}

		std::sort(shortlistedMassSumAndIntensities.begin(), shortlistedMassSumAndIntensities.end(),
			[](const _ShortlistedMassSumsAndIntensities &mass, const _ShortlistedMassSumsAndIntensities &mass2)
		{ return (mass.massSum < mass2.massSum); });

		double minSum = shortlistedMassSumAndIntensities[0].massSum;
		double maxSum = shortlistedMassSumAndIntensities[shortlistedMassSumAndIntensities.size() - 1].massSum;

		int sizeOfShortlistedData = shortlistedMassSumAndIntensities.size();
		double SliderValue = (WholeProteinMass * Parameters.SliderValue) / (pow(10.0, 6.0));
		int NumOfThreadsToLaunch = floor((maxSum - minSum) * (1 / SliderValue));

		thrust::device_vector<_ShortlistedMassSumsAndIntensities> device_shortlistedMassSumAndIntensities = shortlistedMassSumAndIntensities;
		_ShortlistedMassSumsAndIntensities *raw_ptr = thrust::raw_pointer_cast(device_shortlistedMassSumAndIntensities.data());

		thrust::device_vector<_WindowCapturedElementsStruct> device_windowcapturedelements(NumOfThreadsToLaunch);
		_WindowCapturedElementsStruct *raw_ptr2 = thrust::raw_pointer_cast(device_windowcapturedelements.data());

		int THREADS2 = 256;
		int BLOCKS2 = (NumOfThreadsToLaunch / THREADS + 5);

		WindowLaunchKernel << <BLOCKS2, THREADS2 >> > (NumOfThreadsToLaunch, minSum, maxSum, raw_ptr, sizeOfShortlistedData, raw_ptr2, SliderValue);

		thrust::host_vector<_WindowCapturedElementsStruct> host_windowcapturedelements = device_windowcapturedelements;


		int oldElementCount = 0;

		for (int x = 0; x < NumOfThreadsToLaunch; x++)
		{
			if (oldElementCount < host_windowcapturedelements[x].elementCount)
			{
				oldElementCount = host_windowcapturedelements[x].elementCount;
				TunedMass = host_windowcapturedelements[x].TunedMass;
			}
			else if (oldElementCount == host_windowcapturedelements[x].elementCount)
			{
				if (abs(TunedMass - WholeProteinMass) >= abs(host_windowcapturedelements[x].TunedMass - WholeProteinMass))
				{
					TunedMass = host_windowcapturedelements[x].TunedMass;
				}
			}
		}
	}

	else
		TunedMass = 0.0;


	auto MassTunerEnd = chrono::steady_clock::now();
	int ModularMassTunerTime = chrono::duration_cast<chrono::milliseconds>(MassTunerEnd - start).count();


	// --------- PST STARTS HERE ---------
	
	vector<_PeptideSequenceTags> PeptideSequenceTags;

	if (Parameters.DenovoAllow == 1)
	{
		auto PstStart = chrono::steady_clock::now();
		PeptideSequenceTags = CalculatingPeptideSequenceTags(Host_SingleLengthPSTs, Parameters, zN);
		// --------- PST ENDS HERE ---------

		for (int i = 0; i < PeptideSequenceTags.size(); i++) {
			(*_MassTunerAndPstCombinedStruct)->PstTagLength = PeptideSequenceTags[i].PstTagLength;

			(*_MassTunerAndPstCombinedStruct)->PstErrorScore = PeptideSequenceTags[i].PstErrorScore;
			(*_MassTunerAndPstCombinedStruct)->PstFrequency = PeptideSequenceTags[i].PstFrequency;
			(*_MassTunerAndPstCombinedStruct)->MassTuner = TunedMass;

			for (int j = 0; j < PeptideSequenceTags[i].PstTagLength; j++)
			{
				(*_MassTunerAndPstCombinedStruct)->PstTags[j] = PeptideSequenceTags[i].PstTag[j];
			}
			_MassTunerAndPstCombinedStruct++;
		}

		auto PstEnd = chrono::steady_clock::now();
		int ModularPstEndTime = chrono::duration_cast<chrono::milliseconds>(PstEnd - PstStart).count();

		hipDeviceSynchronize();
		return PeptideSequenceTags.size();

	}

	else
	{
		(*_MassTunerAndPstCombinedStruct)->PstTagLength = 0;
		(*_MassTunerAndPstCombinedStruct)->PstErrorScore = 0;
		(*_MassTunerAndPstCombinedStruct)->PstFrequency = 0;
		(*_MassTunerAndPstCombinedStruct)->PstTags[0] = '\0';

		(*_MassTunerAndPstCombinedStruct)->MassTuner = TunedMass;
		
		return 1;
	}
}

