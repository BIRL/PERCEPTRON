#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hipblas.h>
#include <vector>
#include <assert.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include  <device_atomic_functions.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <string>
#include <iterator>
#include<algorithm>
#include <chrono>
#include <math.h>
#include "CudaHeaderFile.cuh"
#include "CudaHeaderForPst.cuh"

#define SIZE	1024 
using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

//	--- Updated: 20210223 ---
//__device__ int dev_data[50];
//__device__ int dev_count = 0;
//__device__ int dev_wind_count = 0;
//__device__ int dev_pst_count = 0;
//__device__ int multipleLengthPstCounter = 0;
//__device__ int charArrayCounter = 0;
//
//struct ParametersToCpp
//{
//	double MwTolerance;
//	double NeutralLoss;
//	double SliderValue;
//	double HopThreshhold;
//	int Autotune;
//	int DenovoAllow;
//	int MinimumPstLength;
//	int MaximumPstLength;
//};
//
//typedef struct _WindowCapturedElementsStruct
//{
//	double TunedMass;
//	int elementCount;
//} windowcapturedelementsstruct;
//
//typedef struct _ShortlistedMassSumsAndIntensities
//{
//	double massSum;
//	double AvgIntensity;
//	bool operator() (_ShortlistedMassSumsAndIntensities i, _ShortlistedMassSumsAndIntensities j) { return (i.massSum < j.massSum); }
//} ShortlistedMassSumsAndIntensities;
//
//typedef struct _DataForPsts
//{
//	int startIndex;
//	int endIndex;
//	double startIndexMass;
//	double endIndexMass;
//	double massDifferenceBetweenPeaks;
//	char AminoAcidSymbol;
//	double TagError;
//	double averageIntensity;
//} dataforpsts;
//
//typedef struct _PeptideSequenceTags
//{
//	char PstTag[8];
//	int PstTagLength;
//	double PstErrorScore;
//	double PstFrequency;
//	double IntensitySum;
//	int startIndex;
//	int endIndex;
//	double ErrorSum;
//	double RMSE;
//} peptidesequencetags;


//__device__ void my_push_back(double *dev_PeakListMassesSum, double *dev_PeakListIntensitiesAverage, double summationOfMasses, double averageOfIntensities)
//{
//	int insert_pt = atomicAdd(&dev_count, 1);
//	dev_PeakListMassesSum[insert_pt] = summationOfMasses;
//	dev_PeakListIntensitiesAverage[insert_pt] = averageOfIntensities;
//	return;
//}

//__device__ void PST_push_back(_DataForPsts *SingleLengthPSTs_ptr, int tid, int i, double StartIndMass, double EndIndMass, double differenceOfMasses, char aminoAcidSymbol, double TagError, double averageOfIntensities)
//{
//	int insert_ptr = atomicAdd(&dev_pst_count, 1);
//	SingleLengthPSTs_ptr[insert_ptr].startIndex = tid;
//	SingleLengthPSTs_ptr[insert_ptr].endIndex = i;
//	SingleLengthPSTs_ptr[insert_ptr].startIndexMass = StartIndMass;
//	SingleLengthPSTs_ptr[insert_ptr].endIndexMass = EndIndMass;
//	SingleLengthPSTs_ptr[insert_ptr].massDifferenceBetweenPeaks = differenceOfMasses;
//	SingleLengthPSTs_ptr[insert_ptr].AminoAcidSymbol = aminoAcidSymbol;
//	SingleLengthPSTs_ptr[insert_ptr].TagError = TagError;
//	SingleLengthPSTs_ptr[insert_ptr].averageIntensity = averageOfIntensities;
//	return;
//}

//__global__ void CalculatingTupleSumsAndSingleLengthPsts(double *raw_ptr_masses, double *raw_ptr_intensities, double *dev_PeakListMassesSum, double *dev_PeakListIntensitiesAverage, _DataForPsts *SingleLengthPSTs_ptr, double *dev_aminoAcidMassesList, char *dev_aminoAcidSymbolList, double MwTolerance, double NeutralLoss, double HopThreshold, int N, int AutoTune, int DenovoAllow) {
//	int tid = blockIdx.x * blockDim.x + threadIdx.x;
//
//	if (tid < N)
//	{
//		for (int i = tid + 1; i < N; i++)
//		{
//			double averageOfIntensities = (raw_ptr_intensities[tid] + raw_ptr_intensities[i]) / 2;
//			if (AutoTune == 1)
//			{
//				double summationOfMasses = raw_ptr_masses[tid] + raw_ptr_masses[i] + NeutralLoss;
//				my_push_back(dev_PeakListMassesSum, dev_PeakListIntensitiesAverage, summationOfMasses, averageOfIntensities);
//			}
//			if (DenovoAllow == 1)
//			{
//				double differenceOfMasses = fabs(raw_ptr_masses[tid] - raw_ptr_masses[i]);
//				for (int j = 0; j < 21; j++)
//				{
//					double TagError = pow(fabs(dev_aminoAcidMassesList[j] - differenceOfMasses), 2);
//					if (fabs(dev_aminoAcidMassesList[j] - differenceOfMasses) <= HopThreshold)
//					{
//						PST_push_back(SingleLengthPSTs_ptr, tid, i, raw_ptr_masses[tid], raw_ptr_masses[i], differenceOfMasses, dev_aminoAcidSymbolList[j], TagError, averageOfIntensities);
//					}
//				}
//			}		
//		}
//	}
//	else
//		return;
//}

//__device__ void window_push_back(_WindowCapturedElementsStruct *windowcapturedelements, double a, int b)
//{
//	int insert_ptr = atomicAdd(&dev_wind_count, 1);
//	windowcapturedelements[insert_ptr].TunedMass = a;
//	windowcapturedelements[insert_ptr].elementCount = b;
//	return;
//}

//__global__ void WindowLaunchKernel(int NumOfThreadsToLaunch, double minSum, double maxSum, _ShortlistedMassSumsAndIntensities *shortListedData, int sizeOfShortlistedData, _WindowCapturedElementsStruct *windowcapturedelements, double SliderValue)
//{
//	int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	if (tid < NumOfThreadsToLaunch)
//	{
//		double WindowStart = minSum + (tid * SliderValue);
//		double WindowEnd = WindowStart + 1.00727647;
//		double sumoftunedmassesandintensities = 0;
//		double sumoftunedintensities = 0;
//		int Count = 0;
//		for (int i = 0; i < sizeOfShortlistedData; i++)
//		{
//			if (WindowStart <= shortListedData[i].massSum && shortListedData[i].massSum < WindowEnd)//#DISCUSSION
//			{
//				double data = shortListedData[i].massSum * shortListedData[i].AvgIntensity;
//				sumoftunedmassesandintensities = sumoftunedmassesandintensities + data;
//				sumoftunedintensities = sumoftunedintensities + shortListedData[i].AvgIntensity;
//				Count = Count + 1;
//			}
//			else if (shortListedData[i].massSum >= WindowEnd)
//			{
//				break;
//			}
//		}
//		double TunedMass = sumoftunedmassesandintensities / sumoftunedintensities;
//		int elementCount = Count;
//		window_push_back(windowcapturedelements, TunedMass, elementCount);
//	}
//}

//__device__ char * my_strcpy(char *dest, const char *src) {
//	int i = 0;
//	do {
//		dest[i] = src[i];
//	} while (src[i++] != 0);
//	return dest;
//}

//__global__ void GeneratingMultipleLengthPsts(_PeptideSequenceTags *MultipleLengthPst_ptr, _DataForPsts *SingleLengthPSTs, int N)
//{
//	int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	if (tid < N)
//	{
//		int HomePeak = SingleLengthPSTs[tid].endIndex;
//		char HomePeakAA = SingleLengthPSTs[tid].AminoAcidSymbol;
//		int StartIndex = SingleLengthPSTs[tid].startIndex;
//		for (int i = 0; i < N; i++)
//		{
//			int HopPeak = SingleLengthPSTs[i].startIndex;
//			char HopPeakAA = SingleLengthPSTs[i].AminoAcidSymbol;
//			int EndIndex = SingleLengthPSTs[i].endIndex;
//			if (HomePeak == HopPeak)
//			{
//				int insert_ptr = atomicAdd(&multipleLengthPstCounter, 1);
//				MultipleLengthPst_ptr[insert_ptr].PstTag[0] = HomePeakAA;
//				MultipleLengthPst_ptr[insert_ptr].PstTag[1] = HopPeakAA;
//				MultipleLengthPst_ptr[insert_ptr].startIndex = StartIndex;
//				MultipleLengthPst_ptr[insert_ptr].endIndex = EndIndex;
//				MultipleLengthPst_ptr[insert_ptr].PstTagLength = 2;
//				MultipleLengthPst_ptr[insert_ptr].IntensitySum = SingleLengthPSTs[tid].averageIntensity + SingleLengthPSTs[i].averageIntensity;
//				MultipleLengthPst_ptr[insert_ptr].PstFrequency = ((SingleLengthPSTs[tid].averageIntensity + SingleLengthPSTs[i].averageIntensity) / 2)*(2*2);
//				MultipleLengthPst_ptr[insert_ptr].ErrorSum = SingleLengthPSTs[tid].TagError + SingleLengthPSTs[i].TagError;
//				MultipleLengthPst_ptr[insert_ptr].RMSE = MultipleLengthPst_ptr[insert_ptr].ErrorSum / 2;
//				double RMSE = (sqrt(MultipleLengthPst_ptr[insert_ptr].ErrorSum) / 2)*10;
//				MultipleLengthPst_ptr[insert_ptr].PstErrorScore = exp(-RMSE * 2);
//				MultipleLengthPst_ptr[insert_ptr].PstErrorScore = (MultipleLengthPst_ptr[insert_ptr].PstTagLength * MultipleLengthPst_ptr[insert_ptr].PstFrequency) / RMSE;
//			}
//		}
//	}
//}

//__global__ void GeneratingMultipleLengthPsts2(_PeptideSequenceTags *MultipleLengthPst_ptr, _PeptideSequenceTags *DupletPSTs, _DataForPsts *SingleLengthPSTs, int N, int num, int SizeOfPst)
//{
//	int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	if (tid < N)
//	{
//		int HomePeak = DupletPSTs[tid].endIndex;
//		char *HomePeakAA = DupletPSTs[tid].PstTag;
//		int StartIndex = DupletPSTs[tid].startIndex;
//		for (int i = 0; i < num; i++)
//		{
//			int HopPeak = SingleLengthPSTs[i].startIndex;
//			char HopPeakAA = SingleLengthPSTs[i].AminoAcidSymbol;
//			int EndIndex = SingleLengthPSTs[i].endIndex;
//			if (HomePeak == HopPeak)
//			{
//				int insert_ptr = atomicAdd(&multipleLengthPstCounter, 1);
//				my_strcpy(MultipleLengthPst_ptr[insert_ptr].PstTag, HomePeakAA);
//				MultipleLengthPst_ptr[insert_ptr].PstTag[SizeOfPst-1] = HopPeakAA;
//				MultipleLengthPst_ptr[insert_ptr].startIndex = StartIndex;
//				MultipleLengthPst_ptr[insert_ptr].endIndex = EndIndex;
//				MultipleLengthPst_ptr[insert_ptr].PstTagLength = SizeOfPst;
//				MultipleLengthPst_ptr[insert_ptr].IntensitySum = DupletPSTs[tid].IntensitySum + SingleLengthPSTs[i].averageIntensity;
//				MultipleLengthPst_ptr[insert_ptr].PstFrequency = ((DupletPSTs[tid].IntensitySum + SingleLengthPSTs[i].averageIntensity) / SizeOfPst)*(SizeOfPst*SizeOfPst);
//				MultipleLengthPst_ptr[insert_ptr].ErrorSum = DupletPSTs[tid].ErrorSum + SingleLengthPSTs[i].TagError;
//				MultipleLengthPst_ptr[insert_ptr].RMSE = (DupletPSTs[tid].ErrorSum + SingleLengthPSTs[i].TagError) / SizeOfPst;
//				double RMSE = (sqrt(DupletPSTs[tid].ErrorSum + SingleLengthPSTs[i].TagError) / SizeOfPst) * 10;
//				MultipleLengthPst_ptr[insert_ptr].PstErrorScore = exp(-RMSE * 2);
//				//MultipleLengthPst_ptr[insert_ptr].PstErrorScore = (MultipleLengthPst_ptr[insert_ptr].PstTagLength * MultipleLengthPst_ptr[insert_ptr].PstFrequency) / RMSE;
//			}
//		}
//	}
//}
void AccomodateIsoforms(vector< _PeptideSequenceTags> &Final_MultipleLengthPsts, ParametersToCpp Parameters)
{
	char ResidueForReplacement[] = { 'L', 'D', 'N', 'E', 'Q' };
	char newResidue;
	int CountOfFinalPsts = Final_MultipleLengthPsts.size();
	int g = strlen(ResidueForReplacement);
	for (int i = 0; i < Final_MultipleLengthPsts.size(); i++)
	{
		for (int j = 0; j < 5; j++)
		{
			char OldResidue = ResidueForReplacement[j];
			if (strchr(Final_MultipleLengthPsts[i].PstTag, OldResidue))
			{
				if (OldResidue == 'L')//Here I think Switch Case will be more better....!!!!
					newResidue = 'I';
				else if (OldResidue == 'D')
					newResidue = 'B';
				else if (OldResidue == 'N')
					newResidue = 'B';
				else if (OldResidue == 'E')
					newResidue = 'Z';
				else if (OldResidue == 'Q' && Parameters.HopThreshhold <= 1.5)
					newResidue = 'Z';
				else if (OldResidue == 'Q' && Parameters.HopThreshhold > 1.5)
					newResidue = 'K';

				string BeforeAccomodatePst = Final_MultipleLengthPsts[i].PstTag;
				_PeptideSequenceTags AccomodatedPstTag = Final_MultipleLengthPsts[i];
				for (int iter = 0; iter < Final_MultipleLengthPsts[i].PstTagLength; iter++)
				{
					if (BeforeAccomodatePst[iter] == OldResidue)
					{
						BeforeAccomodatePst[iter] = newResidue;
						strcpy(AccomodatedPstTag.PstTag, BeforeAccomodatePst.c_str());
						Final_MultipleLengthPsts.push_back(AccomodatedPstTag);
					}
				}
			}
		}
	}
	for (int i = 0; i < Final_MultipleLengthPsts.size() - 1; i++)
	{
		if (strcmp(Final_MultipleLengthPsts[i].PstTag, Final_MultipleLengthPsts[i + 1].PstTag) == 0)
		{
			Final_MultipleLengthPsts.erase(Final_MultipleLengthPsts.begin() + i + 1);
			i--;
		}
	}
}

void FindingUniquePSTs(vector< _PeptideSequenceTags> &Final_MultipleLengthPsts)
{
	for (int i = 0; i < Final_MultipleLengthPsts.size() - 1; i++)	// extracting the unique PSTs from the total PSTs obtained. If 2 PST tags are same, their error sums and intensities are compared
	{
		if (strcmp(Final_MultipleLengthPsts[i].PstTag, Final_MultipleLengthPsts[i + 1].PstTag) == 0)
		{
			if (Final_MultipleLengthPsts[i].ErrorSum < Final_MultipleLengthPsts[i + 1].ErrorSum)
			{
				Final_MultipleLengthPsts.erase(Final_MultipleLengthPsts.begin() + i + 1);
				i--;
			}
			else if (Final_MultipleLengthPsts[i].ErrorSum > Final_MultipleLengthPsts[i + 1].ErrorSum)
			{
				Final_MultipleLengthPsts.erase(Final_MultipleLengthPsts.begin() + i);
				i--;
			}
			else
			{
				if (Final_MultipleLengthPsts[i].PstFrequency <= Final_MultipleLengthPsts[i + 1].PstFrequency)
				{
					Final_MultipleLengthPsts.erase(Final_MultipleLengthPsts.begin() + i);
					i--;
				}
				else if (Final_MultipleLengthPsts[i].PstFrequency > Final_MultipleLengthPsts[i + 1].PstFrequency)
				{
					Final_MultipleLengthPsts.erase(Final_MultipleLengthPsts.begin() + i + 1);
					i--;
				}
			}
		}
	}
}

vector<_PeptideSequenceTags> CalculatingPeptideSequenceTags(thrust::host_vector<_DataForPsts> Host_SingleLengthPSTs, ParametersToCpp Parameters, int zN)
{
	thrust::device_vector<_DataForPsts> Final_SingleLengthPSTs;

	for (int i = 0; i < zN; i++)	// Populating the single length PSTs calculated while running mass tuner function
	{
		if (Host_SingleLengthPSTs[i].startIndexMass == 0)
			break;
		Final_SingleLengthPSTs.push_back(Host_SingleLengthPSTs[i]);
	}
	_DataForPsts *Final_SingleLengthPSTs_ptr = thrust::raw_pointer_cast(Final_SingleLengthPSTs.data());	// Pointer pointing to single length PST vector that is to be sent to gpu

	thrust::device_vector<_PeptideSequenceTags> dev_MultipleLengthPst(1000);
	_PeptideSequenceTags *MultipleLengthPst_ptr = thrust::raw_pointer_cast(dev_MultipleLengthPst.data());	// multiple length PSTs pointer

	int numOfThr = Final_SingleLengthPSTs.size();	// each single length PST will be compared with the others in each thread
	int THREADSforPst = 256;	// optimal num of threads per block
	int BLOCKSforPst = numOfThr / THREADSforPst + 5;	// total threads we want to launch and the optimal threads will give us the num of blocks to be launched. +5 gist foe safety
	GeneratingMultipleLengthPsts << <BLOCKSforPst, THREADSforPst >> > (MultipleLengthPst_ptr, Final_SingleLengthPSTs_ptr, numOfThr);	// Here duplets of PSTs will be created in GPU
	thrust::host_vector<_PeptideSequenceTags> host_MultipleLengthPst = dev_MultipleLengthPst;	// Copying data back to CPU from GPU
	thrust::device_vector<_PeptideSequenceTags> dev_MultipleLengthPst2;

	for (int i = 0; i < host_MultipleLengthPst.size(); i++)	// Extracting the duplets and chopping the excess data so these duplets can be passed to gpu again to compute triplets
	{
		if (host_MultipleLengthPst[i].endIndex == 0)
			break;
		dev_MultipleLengthPst2.push_back(host_MultipleLengthPst[i]);
	}
	_PeptideSequenceTags *MultipleLengthPst_ptr2 = thrust::raw_pointer_cast(dev_MultipleLengthPst2.data());

	int size = 0;
	thrust::host_vector<_PeptideSequenceTags> host_MultipleLengthPst2;

	for (int num = 3; num <= Parameters.MaximumPstLength; num++)	// loop runs to calculate triplets, tetraplets, pentaplets and so on according to user defined parameters
	{
		numOfThr = dev_MultipleLengthPst2.size();
		int PstSize = num;
		GeneratingMultipleLengthPsts2 << <BLOCKSforPst, THREADSforPst >> > (MultipleLengthPst_ptr, MultipleLengthPst_ptr2, Final_SingleLengthPSTs_ptr, numOfThr, Final_SingleLengthPSTs.size(), PstSize);	// gpu kernel to calculate multiple length psts
		host_MultipleLengthPst = dev_MultipleLengthPst;

		size = size + dev_MultipleLengthPst2.size();
		dev_MultipleLengthPst2.clear();
		for (int i = size; i < host_MultipleLengthPst.size(); i++)	// Extracting the PSTs and chopping the excess data so these PSTs can be passed to gpu again to compute further longer length PSTs
		{
			if (host_MultipleLengthPst[i].endIndex == 0)
				break;
			dev_MultipleLengthPst2.push_back(host_MultipleLengthPst[i]);
		}
	}

	vector<_PeptideSequenceTags> Final_MultipleLengthPsts;
	for (int i = 0; i < host_MultipleLengthPst.size(); i++)	// Final multiple length PSTs being stored into vector according to user defined min and max PST length
	{
		if (host_MultipleLengthPst[i].endIndex == 0)
			break;
		if (host_MultipleLengthPst[i].PstTagLength >= Parameters.MinimumPstLength && host_MultipleLengthPst[i].PstTagLength <= Parameters.MaximumPstLength)
		{
			Final_MultipleLengthPsts.push_back(host_MultipleLengthPst[i]);
		}
	}

	std::sort(Final_MultipleLengthPsts.begin(), Final_MultipleLengthPsts.end(),	// sorting the data according to the PST tags
		[](const _PeptideSequenceTags &pst1, const _PeptideSequenceTags &pst2)
	{ return strcmp(pst1.PstTag, pst2.PstTag) < 0; });

	Final_MultipleLengthPsts[0] = Final_MultipleLengthPsts[0];	// DELME Just for checking values
	
	FindingUniquePSTs(Final_MultipleLengthPsts);	// Function call to extract unique PSTs
	
	Final_MultipleLengthPsts[0] = Final_MultipleLengthPsts[0];	// DELME Just for checking values

	AccomodateIsoforms(Final_MultipleLengthPsts, Parameters);	// Isoforms are being accomodated here and then their unique is taken

	Final_MultipleLengthPsts[0] = Final_MultipleLengthPsts[0];	// DELME Just for checking values
	return Final_MultipleLengthPsts;
}

extern "C" __declspec(dllexport) double __cdecl
wholeproteinmasstunerandpst(double PeakListMasses[], double PeakListIntensities[], int PeakListLength, ParametersToCpp Parameters)
{
	double WholeProteinMass = PeakListMasses[0];
	vector<double> masses;  vector<double> intensities;
	for (int i = 0; i < PeakListLength; i++)
	{
		masses.push_back(PeakListMasses[i]);
		intensities.push_back(PeakListIntensities[i]);
	}

	const int N = PeakListLength;
	const int zN = (floor(PeakListLength*PeakListLength) / 2) - (floor(PeakListLength / 2));
	double *dev_masses, *dev_intensities;

	double aminoAcidMassesList[21] = { 57.0214600000000, 71.0371100000000, 87.0320300000000, 97.0527600000000, 99.0684100000000, 101.047680000000, 103.009190000000, 113.084060000000, 114.042930000000, 115.026940000000, 128.058580000000, 128.094960000000, 129.042590000000, 131.040490000000, 137.058910000000, 147.068410000000, 156.101110000000, 163.063330000000, 168.964203000000, 186.079310000000, 255.158295000000 };
	char aminoAcidSymbolList[21] = { 'G', 'A', 'S', 'P', 'V', 'T', 'C', 'L', 'N', 'D', 'Q', 'K', 'E', 'M', 'H', 'F', 'R', 'Y', 'U', 'W', 'O' };
	
	double *dev_aminoAcidMassesList; char *dev_aminoAcidSymbolList;

	hipMalloc((void**)&dev_masses, sizeof(double) * N);
	hipMalloc((void**)&dev_intensities, sizeof(double) * N);
	hipMalloc((void**)&dev_aminoAcidMassesList, sizeof(double)*21);
	hipMalloc((void**)&dev_aminoAcidSymbolList, sizeof(char)*21);

	thrust::device_vector<double> DevicePeakListMassesSum(zN);
	double *DevicePeakListMassesSum_ptr = thrust::raw_pointer_cast(DevicePeakListMassesSum.data());
	thrust::device_vector<double> DevicePeakListAvgIntensities(zN);
	double *DevicePeakListAvgIntensities_ptr = thrust::raw_pointer_cast(DevicePeakListAvgIntensities.data());
	thrust::device_vector<_DataForPsts> DeviceSingleLengthPSTs(zN);
	_DataForPsts *SingleLengthPSTs_ptr = thrust::raw_pointer_cast(DeviceSingleLengthPSTs.data());

	hipMemcpy(dev_masses, PeakListMasses, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_intensities, PeakListIntensities, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_aminoAcidMassesList, aminoAcidMassesList, 21 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_aminoAcidSymbolList, aminoAcidSymbolList, 21 * sizeof(char), hipMemcpyHostToDevice);

	int THREADS = 256;
	int BLOCKS = (N/THREADS + 5);
	CalculatingTupleSumsAndSingleLengthPsts << < BLOCKS, THREADS >> > (dev_masses, dev_intensities, DevicePeakListMassesSum_ptr, DevicePeakListAvgIntensities_ptr, SingleLengthPSTs_ptr, dev_aminoAcidMassesList, dev_aminoAcidSymbolList, Parameters.MwTolerance, Parameters.NeutralLoss, Parameters.HopThreshhold, N, Parameters.Autotune, Parameters.DenovoAllow);

	thrust::host_vector<_DataForPsts> Host_SingleLengthPSTs = DeviceSingleLengthPSTs;
	thrust::host_vector<double> PeakListMassesSum = DevicePeakListMassesSum;
	thrust::host_vector<double> PeakListIntensitiesAverage = DevicePeakListAvgIntensities;
	thrust::host_vector<_ShortlistedMassSumsAndIntensities> shortlistedMassSumAndIntensities;

	for (int i = 0; i < zN; i++)
	{
		if (PeakListMasses[0] - Parameters.MwTolerance <= PeakListMassesSum[i] && PeakListMassesSum[i] <= PeakListMasses[0] + Parameters.MwTolerance)
		{
			_ShortlistedMassSumsAndIntensities data;
			data.massSum = PeakListMassesSum[i];
			data.AvgIntensity = PeakListIntensitiesAverage[i];
			shortlistedMassSumAndIntensities.push_back(data);
		}
	}

	std::sort(shortlistedMassSumAndIntensities.begin(), shortlistedMassSumAndIntensities.end(),
		[](const _ShortlistedMassSumsAndIntensities &mass, const _ShortlistedMassSumsAndIntensities &mass2)
	{ return (mass.massSum < mass2.massSum); });

	double minSum = shortlistedMassSumAndIntensities[0].massSum;
	double maxSum = shortlistedMassSumAndIntensities[shortlistedMassSumAndIntensities.size() - 1].massSum;

	int sizeOfShortlistedData = shortlistedMassSumAndIntensities.size();
	double SliderValue = (WholeProteinMass * Parameters.SliderValue) / (pow(10.0, 6.0));
	int NumOfThreadsToLaunch = floor((maxSum - minSum) * (1 / SliderValue));

	thrust::device_vector<_ShortlistedMassSumsAndIntensities> device_shortlistedMassSumAndIntensities = shortlistedMassSumAndIntensities;
	_ShortlistedMassSumsAndIntensities *raw_ptr = thrust::raw_pointer_cast(device_shortlistedMassSumAndIntensities.data());

	thrust::device_vector<_WindowCapturedElementsStruct> device_windowcapturedelements(NumOfThreadsToLaunch);
	_WindowCapturedElementsStruct *raw_ptr2 = thrust::raw_pointer_cast(device_windowcapturedelements.data());

	int THREADS2 = 256;
	int BLOCKS2 = (NumOfThreadsToLaunch / THREADS + 5);

	WindowLaunchKernel << <BLOCKS2, THREADS2 >> > (NumOfThreadsToLaunch, minSum, maxSum, raw_ptr, sizeOfShortlistedData, raw_ptr2, SliderValue);

	thrust::host_vector<_WindowCapturedElementsStruct> host_windowcapturedelements = device_windowcapturedelements;

	double TunedMass = 0;
	int oldElementCount = 0;

	for (int x = 0; x < NumOfThreadsToLaunch; x++)
	{
		if (oldElementCount < host_windowcapturedelements[x].elementCount)
		{
			oldElementCount = host_windowcapturedelements[x].elementCount;
			TunedMass = host_windowcapturedelements[x].TunedMass;
		}
		else if (oldElementCount == host_windowcapturedelements[x].elementCount)
		{
			if (abs(TunedMass - WholeProteinMass) >= abs(host_windowcapturedelements[x].TunedMass - WholeProteinMass))
			{
				TunedMass = host_windowcapturedelements[x].TunedMass;
			}
		}
	}

	// --------- PST STARTS HERE ---------

	vector<_PeptideSequenceTags> PeptideSequenceTags = CalculatingPeptideSequenceTags(Host_SingleLengthPSTs, Parameters, zN);

	// --------- PST ENDS HERE ---------

	hipDeviceSynchronize();
	return TunedMass;
}
