#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hipblas.h>

#include <assert.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include  <device_atomic_functions.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <string>
#include <iterator>
#include<algorithm>
#include <chrono>
#include <math.h>
#include <vector>
//#include "CudaHeaderFile.cuh"
using namespace std;

__device__ int index_count = 0;

struct ParametersToCpp
{
	double MwTolerance;
	double NeutralLoss;
	double SliderValue;
	double HopThreshhold;
	int Autotune;
	int DenovoAllow;
	int MinimumPstLength;
	int MaximumPstLength;
	const char* peptideToleranceUnit;
	double peptideTolerance;
};

struct ProteinStructFromCS
{
	const char* Header;
	double* InsilicoMassLeft;
	double* InsilicoMassRight;
	double* InsilicoMassLeftAo;
	double* InsilicoMassLeftBo;
	double* InsilicoMassLeftAstar;
	double* InsilicoMassLeftBstar;
	double* InsilicoMassRightYo;
	double* InsilicoMassRightYstar;
	double* InsilicoMassRightZo;
	double* InsilicoMassRightZoo;
	int* SizeOfAllInsilicoArrays;
};
struct ProteinStructToReturn
{
	int Header;
	int MatchCounter;
	double InsilicoScore;
	int* LeftMatchedIndex;
	int* RightMatchedIndex;
	int* LeftPeakIndex;
	int* RightPeakIndex;
	int* LeftType;
	int* RightType;
	int LeftCount;
	int RightCount;
};

typedef struct ToDefineSizeStruct
{
	int sizeOfArray;
}Todefinesizestruct;

int PeptideTolUnitMapping(string pepUnit)
{
	int PepUnit;
	if (pepUnit == "Da")
		return PepUnit = 1;
	if (pepUnit == "mmu")
		return PepUnit = 2;
	if (pepUnit == "ppm")
		return PepUnit = 3;
}



__device__ void SpectralComparison(double difference, double dev_intensity, int indexPeakList, double peakDifferenceTolerance, int &Consecutive, int &Counter,
	int &OldConsec, int &OldConsec2, int &ConsecutiveRegion, double &Matches_Score, int &MatchCounter, int *Matched_IndexList, int *Peak_IndexList,
	int indexSide, int Type, int *TypeList, int &insert_ptr)
{
	double absdifference = fabs(difference);  //Taking Absoulte difference {Doesn't matter}
	int *acfg;
	if (absdifference <= peakDifferenceTolerance)
	{
		if (Consecutive == OldConsec + 1 && OldConsec == OldConsec2 + 1)
		{
			if (Counter == 0)
			{
				ConsecutiveRegion = ConsecutiveRegion + 1;
			}
			Counter = Counter + 1;
			Matches_Score = Matches_Score + 1.5;
			OldConsec2 = OldConsec;
			OldConsec = Consecutive;
		}
		else if (Consecutive == OldConsec && OldConsec == OldConsec2 + 1)
		{
			Counter = Counter + 1;
			Matches_Score = Matches_Score + 1.5;
		}
		else
		{
			Counter = 0;
			Matches_Score = Matches_Score + dev_intensity;
			OldConsec2 = OldConsec;
			OldConsec = Consecutive;
		}
		//int insert_ptr = atomicAdd(&index_count, 1);
		Matched_IndexList[insert_ptr] = indexSide;
		Peak_IndexList[insert_ptr] = indexPeakList;
		TypeList[insert_ptr] = Type;
		MatchCounter = MatchCounter + 1;
		insert_ptr = insert_ptr + 1;
	}
}



__global__ void ComputeInsilicoScore(ProteinStructFromCS *h_a, ProteinStructToReturn *DeviceCandidateProteinReturnPtr,
	int candidateProteinsCount, ParametersToCpp Parameters, double *dev_masses, double *dev_intensities, int PeakListCount, int intPepUnit)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < candidateProteinsCount)
	{
		ProteinStructFromCS temp = h_a[tid];

		int Type;
		int InsilicoMassLeftCount = temp.SizeOfAllInsilicoArrays[0], InsilicoMassRightCount = temp.SizeOfAllInsilicoArrays[1],
			InsilicoMassLeftAoCount = temp.SizeOfAllInsilicoArrays[2], InsilicoMassLeftBoCount = temp.SizeOfAllInsilicoArrays[3],
			InsilicoMassLeftAstarCount = temp.SizeOfAllInsilicoArrays[4], InsilicoMassLeftBstarCount = temp.SizeOfAllInsilicoArrays[5],
			InsilicoMassRightYoCount = temp.SizeOfAllInsilicoArrays[6], InsilicoMassRightYstarCount = temp.SizeOfAllInsilicoArrays[7],
			InsilicoMassRightZoCount = temp.SizeOfAllInsilicoArrays[8], InsilicoMassRightZooCount = temp.SizeOfAllInsilicoArrays[9];


		int SpecialLeftFragments = temp.SizeOfAllInsilicoArrays[0] + temp.SizeOfAllInsilicoArrays[2] + temp.SizeOfAllInsilicoArrays[3] +
			temp.SizeOfAllInsilicoArrays[4] + temp.SizeOfAllInsilicoArrays[5];
		int SpecialRightFragments = temp.SizeOfAllInsilicoArrays[1] + temp.SizeOfAllInsilicoArrays[6] + temp.SizeOfAllInsilicoArrays[7] +
			temp.SizeOfAllInsilicoArrays[8] + temp.SizeOfAllInsilicoArrays[9];

		double Matches_Score = 0;
		int MatchCounter = 0;
		int Counter = 0;
		int OldConsec = -1;
		int OldConsec2 = -1;
		int ConsecutiveRegion = 0;
		int IdxL = 0;
		int IdxR = 0;
		int LeftMatched_Index[5000];
		int LeftPeak_Index[5000];
		int LeftType[5000];
		int RightMatched_Index[5000];
		int RightPeak_Index[5000];
		int RightType[5000];
		int insert_ptrLeft = 0;
		int insert_ptrRight = 0;
		double a = 0.0;
		double b = 0.0;

		for (int indexPeakList = 1; indexPeakList < PeakListCount; indexPeakList++)//indexPeakList = 1; indexPeakList < PeakListCount; indexPeakList++)
		{
			double peakDifferenceTolerance;
			if (intPepUnit == 1 || intPepUnit == 2)
				peakDifferenceTolerance = Parameters.peptideTolerance;
			else if (intPepUnit == 3)
				peakDifferenceTolerance = (Parameters.peptideTolerance * dev_masses[indexPeakList]) / 1000000;
			int Consecutive = indexPeakList;
			for (int indexLeftSide = IdxL; indexLeftSide < InsilicoMassLeftCount; indexLeftSide++) //indexLeftSide = IdxL; indexLeftSide < InsilicoMassLeftCount; indexLeftSide++)	//indexLeftSide = 27; indexLeftSide < 28; indexLeftSide++) 
			{
				Type = 1;
				a = dev_masses[indexPeakList];
				b = temp.InsilicoMassLeft[indexLeftSide];
				double difference = dev_masses[indexPeakList] - temp.InsilicoMassLeft[indexLeftSide];
				SpectralComparison(difference, dev_intensities[indexPeakList], indexPeakList, peakDifferenceTolerance, Consecutive, Counter, OldConsec, OldConsec2, ConsecutiveRegion, Matches_Score, MatchCounter, LeftMatched_Index, LeftPeak_Index, indexLeftSide, Type, LeftType, insert_ptrLeft);

				if (SpecialLeftFragments > 0)
				{
					if (InsilicoMassLeftAoCount > 0)
					{
						Type = 2;
						difference = dev_masses[indexPeakList] - temp.InsilicoMassLeftAo[indexLeftSide];
						SpectralComparison(difference, dev_intensities[indexPeakList], indexPeakList, peakDifferenceTolerance, Consecutive, Counter, OldConsec, OldConsec2, ConsecutiveRegion, Matches_Score, MatchCounter, LeftMatched_Index, LeftPeak_Index, indexLeftSide, Type, LeftType, insert_ptrLeft);
					}
					if (InsilicoMassLeftBoCount > 0)
					{
						Type = 3;
						difference = dev_masses[indexPeakList] - temp.InsilicoMassLeftBo[indexLeftSide];
						SpectralComparison(difference, dev_intensities[indexPeakList], indexPeakList, peakDifferenceTolerance, Consecutive, Counter, OldConsec, OldConsec2, ConsecutiveRegion, Matches_Score, MatchCounter, LeftMatched_Index, LeftPeak_Index, indexLeftSide, Type, LeftType, insert_ptrLeft);
					}
					if (InsilicoMassLeftAstarCount > 0)
					{
						Type = 4;
						difference = dev_masses[indexPeakList] - temp.InsilicoMassLeftAstar[indexLeftSide];
						SpectralComparison(difference, dev_intensities[indexPeakList], indexPeakList, peakDifferenceTolerance, Consecutive, Counter, OldConsec, OldConsec2, ConsecutiveRegion, Matches_Score, MatchCounter, LeftMatched_Index, LeftPeak_Index, indexLeftSide, Type, LeftType, insert_ptrLeft);
					}
					if (InsilicoMassLeftBstarCount > 0)
					{
						Type = 5;
						difference = dev_masses[indexPeakList] - temp.InsilicoMassLeftBstar[indexLeftSide];
						SpectralComparison(difference, dev_intensities[indexPeakList], indexPeakList, peakDifferenceTolerance, Consecutive, Counter, OldConsec, OldConsec2, ConsecutiveRegion, Matches_Score, MatchCounter, LeftMatched_Index, LeftPeak_Index, indexLeftSide, Type, LeftType, insert_ptrLeft);
						//insert_ptr++;
					}
				}
				if (difference < -peakDifferenceTolerance && indexLeftSide > 0)  // Updated 20200917   -- Changed from 1 to 0
				{
					IdxL = indexLeftSide - 1;
					break;
				}
			}


			for (int indexRightSide = IdxR; indexRightSide < InsilicoMassLeftCount; indexRightSide++)
			{
				Type = 6;
				double difference = dev_masses[indexPeakList] - temp.InsilicoMassRight[indexRightSide];
				SpectralComparison(difference, dev_intensities[indexPeakList], indexPeakList, peakDifferenceTolerance, Consecutive, Counter, OldConsec, OldConsec2, ConsecutiveRegion, Matches_Score, MatchCounter, RightMatched_Index, RightPeak_Index, indexRightSide, Type, RightType, insert_ptrRight);
				if (SpecialLeftFragments > 0)
				{
					if (InsilicoMassRightYoCount > 0)
					{
						Type = 7;
						difference = dev_masses[indexPeakList] - temp.InsilicoMassRightYo[indexRightSide];
						SpectralComparison(difference, dev_intensities[indexPeakList], indexPeakList, peakDifferenceTolerance, Consecutive, Counter, OldConsec, OldConsec2, ConsecutiveRegion, Matches_Score, MatchCounter, RightMatched_Index, RightPeak_Index, indexRightSide, Type, RightType, insert_ptrRight);
					}
					if (InsilicoMassRightYstarCount > 0)
					{
						Type = 8;
						difference = dev_masses[indexPeakList] - temp.InsilicoMassRightYstar[indexRightSide];
						SpectralComparison(difference, dev_intensities[indexPeakList], indexPeakList, peakDifferenceTolerance, Consecutive, Counter, OldConsec, OldConsec2, ConsecutiveRegion, Matches_Score, MatchCounter, RightMatched_Index, RightPeak_Index, indexRightSide, Type, RightType, insert_ptrRight);
					}
					if (InsilicoMassRightZoCount > 0)
					{
						Type = 9;
						difference = dev_masses[indexPeakList] - temp.InsilicoMassRightZo[indexRightSide];
						SpectralComparison(difference, dev_intensities[indexPeakList], indexPeakList, peakDifferenceTolerance, Consecutive, Counter, OldConsec, OldConsec2, ConsecutiveRegion, Matches_Score, MatchCounter, RightMatched_Index, RightPeak_Index, indexRightSide, Type, RightType, insert_ptrRight);
					}
					if (InsilicoMassRightZooCount > 0)
					{
						Type = 10;
						difference = dev_masses[indexPeakList] - temp.InsilicoMassRightZoo[indexRightSide];
						SpectralComparison(difference, dev_intensities[indexPeakList], indexPeakList, peakDifferenceTolerance, Consecutive, Counter, OldConsec, OldConsec2, ConsecutiveRegion, Matches_Score, MatchCounter, RightMatched_Index, RightPeak_Index, indexRightSide, Type, RightType, insert_ptrRight);
					}
				}
				if (difference < -peakDifferenceTolerance && indexRightSide > 0)
				{
					IdxR = indexRightSide - 1;
					break;
				}
			}
		}

		DeviceCandidateProteinReturnPtr[tid].Header = tid;
		DeviceCandidateProteinReturnPtr[tid].InsilicoScore = Matches_Score / PeakListCount;
		DeviceCandidateProteinReturnPtr[tid].MatchCounter = MatchCounter;

		for (int i = 0; i < insert_ptrLeft; i++)
		{
			DeviceCandidateProteinReturnPtr[tid].LeftMatchedIndex[i] = LeftMatched_Index[i];
			DeviceCandidateProteinReturnPtr[tid].LeftPeakIndex[i] = LeftPeak_Index[i];
			DeviceCandidateProteinReturnPtr[tid].LeftType[i] = LeftType[i];
		}

		for (int i = 0; i < insert_ptrRight; i++)
		{
			DeviceCandidateProteinReturnPtr[tid].RightMatchedIndex[i] = RightMatched_Index[i];
			DeviceCandidateProteinReturnPtr[tid].RightPeakIndex[i] = RightPeak_Index[i];
			DeviceCandidateProteinReturnPtr[tid].RightType[i] = RightType[i];
		}

		DeviceCandidateProteinReturnPtr[tid].LeftCount = insert_ptrLeft;
		DeviceCandidateProteinReturnPtr[tid].RightCount = insert_ptrRight;
	}

}

extern "C" __declspec(dllexport) int __cdecl
insilicospectralcomparisongpu(ParametersToCpp Parameters, ProteinStructFromCS **candidateProteins, double *PeakListMasses,
	double *PeakListIntensities, int PeakListCount, int candidateProteinsCount, ProteinStructToReturn **DataToReturn)
{
	ProteinStructFromCS *CandidateProteinsToCuda = new ProteinStructFromCS[candidateProteinsCount];
	ProteinStructFromCS *dev_CandidateProteinsToCuda;

	vector<double> ghk;
	for (int i = 0; i < PeakListCount; i++)
	{
		ghk.push_back(PeakListMasses[i]);
	}


	thrust::host_vector<ProteinStructFromCS> host_CandidateProteinsToCuda;
	ProteinStructFromCS *h_a = new ProteinStructFromCS[candidateProteinsCount];
	ProteinStructFromCS *d_a;
	int *SizeOfAllInsilicoArrays = new int[10];

	//
	int SizeOfLeftIonsTotal = 0;
	int *SizeOfIndividualLeftIons = new int[candidateProteinsCount];
	//
	int SizeOfRightIonsTotal = 0;
	int *SizeOfIndividualRightIons = new int[candidateProteinsCount];

	int SizeOfLeftAoIonsTotal = 0;
	int *SizeOfIndividualLeftAoIons = new int[candidateProteinsCount];

	int SizeOfLeftBoIonsTotal = 0;
	int *SizeOfIndividualLeftBoIons = new int[candidateProteinsCount];

	int SizeOfLeftAstarIonsTotal = 0;
	int *SizeOfIndividualLeftAstarIons = new int[candidateProteinsCount];


	int SizeOfLeftBstarIonsTotal = 0;
	int *SizeOfIndividualLeftBstarIons = new int[candidateProteinsCount];


	int SizeOfRightYoIonsTotal = 0;
	int *SizeOfIndividualRightYoIons = new int[candidateProteinsCount];

	int SizeOfRightYstarIonsTotal = 0;
	int *SizeOfIndividualRightYstarIons = new int[candidateProteinsCount];

	int SizeOfRightZoIonsTotal = 0;
	int *SizeOfIndividualRightZoIons = new int[candidateProteinsCount];

	int SizeOfRightZooIonsTotal = 0;
	int *SizeOfIndividualRightZooIons = new int[candidateProteinsCount];


	for (int i = 0; i < candidateProteinsCount; i++)
	{
		SizeOfAllInsilicoArrays = (*candidateProteins)->SizeOfAllInsilicoArrays;

		CandidateProteinsToCuda[i] = (**candidateProteins);
		SizeOfIndividualLeftIons[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[0];
		SizeOfLeftIonsTotal = SizeOfLeftIonsTotal + (*candidateProteins)->SizeOfAllInsilicoArrays[0];

		SizeOfIndividualRightIons[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[1];
		SizeOfRightIonsTotal = SizeOfRightIonsTotal + (*candidateProteins)->SizeOfAllInsilicoArrays[1];

		SizeOfIndividualLeftAoIons[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[2];
		SizeOfLeftAoIonsTotal = SizeOfLeftAoIonsTotal + (*candidateProteins)->SizeOfAllInsilicoArrays[2];

		SizeOfIndividualLeftBoIons[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[3];
		SizeOfLeftBoIonsTotal = SizeOfLeftBoIonsTotal + (*candidateProteins)->SizeOfAllInsilicoArrays[3];

		SizeOfIndividualLeftAstarIons[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[4];
		SizeOfLeftAstarIonsTotal = SizeOfLeftAstarIonsTotal + (*candidateProteins)->SizeOfAllInsilicoArrays[4];




		SizeOfIndividualLeftBstarIons[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[5];
		SizeOfLeftBstarIonsTotal = SizeOfLeftBstarIonsTotal + (*candidateProteins)->SizeOfAllInsilicoArrays[5];

		SizeOfIndividualRightYoIons[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[6];
		SizeOfRightYoIonsTotal = SizeOfRightYoIonsTotal + (*candidateProteins)->SizeOfAllInsilicoArrays[6];

		SizeOfIndividualRightYstarIons[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[7];
		SizeOfRightYstarIonsTotal = SizeOfRightYstarIonsTotal + (*candidateProteins)->SizeOfAllInsilicoArrays[7];

		SizeOfIndividualRightZoIons[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[8];
		SizeOfRightZoIonsTotal = SizeOfRightZoIonsTotal + (*candidateProteins)->SizeOfAllInsilicoArrays[8];


		SizeOfIndividualRightZooIons[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[9];
		SizeOfRightZooIonsTotal = SizeOfRightZooIonsTotal + (*candidateProteins)->SizeOfAllInsilicoArrays[9];

		//////////////////////////////////////////////////////////////////////
		//SizeOfIndividualLeftMatchedIndex[i] = 5000;
		//SizeOfIndividualRightMatchedIndex[i] = 5000;
		//SizeOfIndividualLeftPeakIndex[i] = 5000;
		//SizeOfIndividualRightPeakIndex[i] = 5000;
		//SizeOfIndividualLeftType[i] = 5000;
		//SizeOfIndividualRightType[i] = 5000;
		//////////////////////////////////////////////////////////////////


		host_CandidateProteinsToCuda.push_back(**candidateProteins);

		candidateProteins++;
	}


	double* h_arr = new double[SizeOfLeftIonsTotal];

	double* h_arr_Right = new double[SizeOfRightIonsTotal];
	double* h_arr_LeftAo = new double[SizeOfLeftAoIonsTotal];
	double* h_arr_LeftBo = new double[SizeOfLeftBoIonsTotal];
	double* h_arr_LeftAstar = new double[SizeOfLeftAstarIonsTotal];
	double* h_arr_LeftBstar = new double[SizeOfLeftBstarIonsTotal];
	double* h_arr_RightYo = new double[SizeOfRightYoIonsTotal];
	double* h_arr_RightYstar = new double[SizeOfRightYstarIonsTotal];
	double* h_arr_RightZo = new double[SizeOfRightZoIonsTotal];
	double* h_arr_RightZoo = new double[SizeOfRightZooIonsTotal];

	int sizeOfSizesArrayTotal = 10 * candidateProteinsCount;
	int *h_arrSizes = new int[sizeOfSizesArrayTotal];

	int k = 0, kRight = 0, kLeftAo = 0, kLeftBo = 0, kLeftAstar = 0, kLeftBstar = 0,
		kRightYo = 0, kRightYstar = 0, kRightZo = 0, kRightZoo = 0;

	int kSizes = 0;
	for (int i = 0; i < candidateProteinsCount; i++)
	{
		for (int j = 0; j < 10; j++)
		{
			h_arrSizes[kSizes] = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[j];
			kSizes++;
		}

		///Left
		int InnerCount = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[0];
		for (int j = 0; j < InnerCount; j++)
		{
			h_arr[k] = host_CandidateProteinsToCuda[i].InsilicoMassLeft[j];
			k++;
		}
		////

		int InnerCountRight = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[1];
		for (int j = 0; j < InnerCountRight; j++)
		{
			h_arr_Right[kRight] = host_CandidateProteinsToCuda[i].InsilicoMassRight[j];
			kRight++;
		}


		int InnerCountLeftAo = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[2];
		for (int j = 0; j < InnerCountLeftAo; j++)
		{
			h_arr_LeftAo[kLeftAo] = host_CandidateProteinsToCuda[i].InsilicoMassLeftAo[j];
			kLeftAo++;
		}


		int InnerCountLeftBo = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[3];
		for (int j = 0; j < InnerCountLeftBo; j++)
		{
			h_arr_LeftBo[kLeftBo] = host_CandidateProteinsToCuda[i].InsilicoMassLeftBo[j];
			kLeftBo++;
		}


		int InnerCountLeftAstar = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[4];
		for (int j = 0; j < InnerCountLeftAstar; j++)
		{
			h_arr_LeftAstar[kLeftAstar] = host_CandidateProteinsToCuda[i].InsilicoMassLeftAstar[j];
			kLeftAstar++;
		}


		int InnerCountLeftBstar = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[5];
		for (int j = 0; j < InnerCountLeftBstar; j++)
		{
			h_arr_LeftBstar[kLeftBstar] = host_CandidateProteinsToCuda[i].InsilicoMassLeftBstar[j];
			kLeftBstar++;
		}

		int InnerCountRightYo = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[6];
		for (int j = 0; j < InnerCountRightYo; j++)
		{
			h_arr_RightYo[kRightYo] = host_CandidateProteinsToCuda[i].InsilicoMassRightYo[j];
			kRightYo++;
		}


		int InnerCountRightYstar = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[7];
		for (int j = 0; j < InnerCountRightYstar; j++)
		{
			h_arr_RightYstar[kRightYstar] = host_CandidateProteinsToCuda[i].InsilicoMassRightYstar[j];
			kRightYstar++;
		}


		int InnerCountRightZo = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[8];
		for (int j = 0; j < InnerCountRightZo; j++)
		{
			h_arr_RightZo[kRightZo] = host_CandidateProteinsToCuda[i].InsilicoMassRightZo[j];
			kRightZo++;
		}


		int InnerCountRightZoo = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[9];
		for (int j = 0; j < InnerCountRightZoo; j++)
		{
			h_arr_RightZoo[kRightZoo] = host_CandidateProteinsToCuda[i].InsilicoMassRightZoo[j];
			kRightZoo++;
		}
	}

	double *d_arr, *d_arrRight, *d_arrLeftAo, *d_arrLeftBo, *d_arrLeftAstar, *d_arrLeftBstar,
		*d_arrRightYo, *d_arrRightYstar, *d_arrRightZo, *d_arrRightZoo;
	int *d_arrSizes;

	hipMalloc((void**) &(d_arrSizes), sizeof(int)*sizeOfSizesArrayTotal);
	hipMemcpy(d_arrSizes, h_arrSizes, sizeof(int)*sizeOfSizesArrayTotal, hipMemcpyHostToDevice);

	hipMalloc((void**) &(d_arr), sizeof(double)*SizeOfLeftIonsTotal);
	hipMemcpy(d_arr, h_arr, sizeof(double)*SizeOfLeftIonsTotal, hipMemcpyHostToDevice);
	//


	hipMalloc((void**) &(d_arrRight), sizeof(double)*SizeOfRightIonsTotal);
	hipMemcpy(d_arrRight, h_arr_Right, sizeof(double)*SizeOfRightIonsTotal, hipMemcpyHostToDevice);


	hipMalloc((void**) &(d_arrLeftAo), sizeof(double)*SizeOfLeftAoIonsTotal);
	hipMemcpy(d_arrLeftAo, h_arr_LeftAo, sizeof(double)*SizeOfLeftAoIonsTotal, hipMemcpyHostToDevice);

	hipMalloc((void**) &(d_arrLeftBo), sizeof(double)*SizeOfLeftBoIonsTotal);
	hipMemcpy(d_arrLeftBo, h_arr_LeftBo, sizeof(double)*SizeOfLeftBoIonsTotal, hipMemcpyHostToDevice);

	hipMalloc((void**) &(d_arrLeftAstar), sizeof(double)*SizeOfLeftAstarIonsTotal);
	hipMemcpy(d_arrLeftAstar, h_arr_LeftAstar, sizeof(double)*SizeOfLeftAstarIonsTotal, hipMemcpyHostToDevice);

	hipMalloc((void**) &(d_arrLeftBstar), sizeof(double)*SizeOfLeftBstarIonsTotal);
	hipMemcpy(d_arrLeftBstar, h_arr_LeftBstar, sizeof(double)*SizeOfLeftBstarIonsTotal, hipMemcpyHostToDevice);

	hipMalloc((void**) &(d_arrRightYo), sizeof(double)*SizeOfRightYoIonsTotal);
	hipMemcpy(d_arrRightYo, h_arr_RightYo, sizeof(double)*SizeOfRightYoIonsTotal, hipMemcpyHostToDevice);

	hipMalloc((void**) &(d_arrRightYstar), sizeof(double)*SizeOfRightYstarIonsTotal);
	hipMemcpy(d_arrRightYstar, h_arr_RightYstar, sizeof(double)*SizeOfRightYstarIonsTotal, hipMemcpyHostToDevice);

	hipMalloc((void**) &(d_arrRightZo), sizeof(double)*SizeOfRightZoIonsTotal);
	hipMemcpy(d_arrRightZo, h_arr_RightZo, sizeof(double)*SizeOfRightZoIonsTotal, hipMemcpyHostToDevice);

	hipMalloc((void**) &(d_arrRightZoo), sizeof(double)*SizeOfRightZooIonsTotal);
	hipMemcpy(d_arrRightZoo, h_arr_RightZoo, sizeof(double)*SizeOfRightZooIonsTotal, hipMemcpyHostToDevice);

	int megaSize = candidateProteinsCount * 5000;

	int *h_LeftMatchedIndex = new int[megaSize], *h_RightMatchedIndex = new int[megaSize], *h_LeftPeakIndex = new int[megaSize], *h_RightPeakIndex = new int[megaSize], *h_LeftType = new int[megaSize], *h_RightType = new int[megaSize];
	for (int i = 0; i < megaSize; i++)
	{
		h_LeftMatchedIndex[i] = 0;
		h_RightMatchedIndex[i] = 0;
		h_LeftPeakIndex[i] = 0;
		h_RightPeakIndex[i] = 0;
		h_LeftType[i] = 0;
		h_RightType[i] = 0;
	}


	int *d_LeftMatchedIndex, *d_RightMatchedIndex, *d_LeftPeakIndex, *d_RightPeakIndex, *d_LeftType, *d_RightType;
	hipMalloc((void**) &(d_LeftMatchedIndex), sizeof(int)*megaSize);
	hipMemcpy(d_LeftMatchedIndex, h_LeftMatchedIndex, sizeof(int)*megaSize, hipMemcpyHostToDevice);
	hipMalloc((void**) &(d_RightMatchedIndex), sizeof(int)*megaSize);
	hipMemcpy(d_RightMatchedIndex, h_RightMatchedIndex, sizeof(int)*megaSize, hipMemcpyHostToDevice);
	hipMalloc((void**) &(d_LeftPeakIndex), sizeof(int)*megaSize);
	hipMemcpy(d_LeftPeakIndex, h_LeftPeakIndex, sizeof(int)*megaSize, hipMemcpyHostToDevice);
	hipMalloc((void**) &(d_RightPeakIndex), sizeof(int)*megaSize);
	hipMemcpy(d_RightPeakIndex, h_RightPeakIndex, sizeof(int)*megaSize, hipMemcpyHostToDevice);
	hipMalloc((void**) &(d_LeftType), sizeof(int)*megaSize);
	hipMemcpy(d_LeftType, h_LeftType, sizeof(int)*megaSize, hipMemcpyHostToDevice);
	hipMalloc((void**) &(d_RightType), sizeof(int)*megaSize);
	hipMemcpy(d_RightType, h_RightType, sizeof(int)*megaSize, hipMemcpyHostToDevice);

	ProteinStructToReturn *h_return = new ProteinStructToReturn[candidateProteinsCount];
	ProteinStructToReturn *d_return;
	///////////////////////////////////////////////////////////////

	int PrevLeftIons = 0, PrevRightIons = 0, PrevLeftAoIons = 0, PrevLeftBoIons = 0,
		PrevLeftAstarIons = 0, PrevLeftBstarIons = 0,
		PrevRightYoIons = 0, PrevRightYstarIons = 0, PrevRightZoIons = 0, PrevRightZooIons = 0;

	for (int i = 0; i < candidateProteinsCount; i++)
	{
		//////////////////////////////////////////////////////////////
		h_return[i].LeftMatchedIndex = d_LeftMatchedIndex + (i * 5000);
		h_return[i].LeftPeakIndex = d_LeftPeakIndex + (i * 5000);
		h_return[i].LeftType = d_LeftType + (i * 5000);
		h_return[i].RightMatchedIndex = d_RightMatchedIndex + (i * 5000);
		h_return[i].RightPeakIndex = d_RightPeakIndex + (i * 5000);
		h_return[i].RightType = d_RightType + (i * 5000);
		////////////////////////////////////////////////////////////

		h_a[i].SizeOfAllInsilicoArrays = d_arrSizes + (i * 10);

		h_a[i].InsilicoMassLeft = d_arr + (PrevLeftIons);
		PrevLeftIons = PrevLeftIons + SizeOfIndividualLeftIons[i];
		///

		h_a[i].InsilicoMassRight = d_arrRight + (PrevRightIons);
		PrevRightIons = PrevRightIons + SizeOfIndividualRightIons[i];

		h_a[i].InsilicoMassLeftAo = d_arrLeftAo + (PrevLeftAoIons);
		PrevLeftAoIons = PrevLeftAoIons + SizeOfIndividualLeftAoIons[i];

		h_a[i].InsilicoMassLeftBo = d_arrLeftBo + (PrevLeftBoIons);
		PrevLeftBoIons = PrevLeftBoIons + SizeOfIndividualLeftBoIons[i];

		h_a[i].InsilicoMassLeftAstar = d_arrLeftAstar + (PrevLeftAstarIons);
		PrevLeftAstarIons = PrevLeftAstarIons + SizeOfIndividualLeftAstarIons[i];

		h_a[i].InsilicoMassLeftBstar = d_arrLeftBstar + (PrevLeftBstarIons);
		PrevLeftBstarIons = PrevLeftBstarIons + SizeOfIndividualLeftBstarIons[i];

		h_a[i].InsilicoMassRightYo = d_arrRightYo + (PrevRightYoIons);
		PrevRightYoIons = PrevRightYoIons + SizeOfIndividualRightYoIons[i];

		h_a[i].InsilicoMassRightYstar = d_arrRightYstar + (PrevRightYstarIons);
		PrevRightYstarIons = PrevRightYstarIons + SizeOfIndividualRightYstarIons[i];

		h_a[i].InsilicoMassRightZo = d_arrRightZo + (PrevRightZoIons);
		PrevRightZoIons = PrevRightZoIons + SizeOfIndividualRightZoIons[i];

		h_a[i].InsilicoMassRightZoo = d_arrRightZoo + (PrevRightZooIons);
		PrevRightZooIons = PrevRightZooIons + SizeOfIndividualRightZooIons[i];

	}

	hipMalloc((void**) &(d_a), sizeof(ProteinStructFromCS)*candidateProteinsCount);
	hipMemcpy(d_a, h_a, sizeof(ProteinStructFromCS)*candidateProteinsCount, hipMemcpyHostToDevice);

	hipMalloc((void**) &(d_return), sizeof(ProteinStructToReturn)*candidateProteinsCount);
	hipMemcpy(d_return, h_return, sizeof(ProteinStructToReturn)*candidateProteinsCount, hipMemcpyHostToDevice);

	int NumOfThreadsToLaunch = 256;
	int NoOfBlocks = candidateProteinsCount / NumOfThreadsToLaunch;	///HERE	

	if (NoOfBlocks == 0)
		NoOfBlocks = 1;

	std::string stringPepUnit = Parameters.peptideToleranceUnit;
	int intPepUnit = PeptideTolUnitMapping(stringPepUnit);

	double *dev_masses, *dev_intensities;
	hipMalloc((void**)&dev_masses, sizeof(double) * PeakListCount);
	hipMalloc((void**)&dev_intensities, sizeof(double) * PeakListCount);
	hipMemcpy(dev_masses, PeakListMasses, PeakListCount * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_intensities, PeakListIntensities, PeakListCount * sizeof(double), hipMemcpyHostToDevice);

	ComputeInsilicoScore << <NoOfBlocks, NumOfThreadsToLaunch >> > (d_a, d_return, candidateProteinsCount, Parameters, dev_masses, dev_intensities, PeakListCount, intPepUnit);

	hipMemcpy(h_a, d_a, sizeof(ProteinStructFromCS)*candidateProteinsCount, hipMemcpyDeviceToHost);
	hipMemcpy(h_arr, d_arr, sizeof(double) * SizeOfLeftIonsTotal, hipMemcpyDeviceToHost);
	hipMemcpy(h_return, d_return, sizeof(ProteinStructToReturn)*candidateProteinsCount, hipMemcpyDeviceToHost);
	hipMemcpy(h_LeftMatchedIndex, d_LeftMatchedIndex, sizeof(int) * megaSize, hipMemcpyDeviceToHost);
	hipMemcpy(h_LeftPeakIndex, d_LeftPeakIndex, sizeof(int) * megaSize, hipMemcpyDeviceToHost);
	hipMemcpy(h_LeftType, d_LeftType, sizeof(int) * megaSize, hipMemcpyDeviceToHost);
	hipMemcpy(h_RightMatchedIndex, d_RightMatchedIndex, sizeof(int) * megaSize, hipMemcpyDeviceToHost);
	hipMemcpy(h_RightPeakIndex, d_RightPeakIndex, sizeof(int) * megaSize, hipMemcpyDeviceToHost);
	hipMemcpy(h_RightType, d_RightType, sizeof(int) * megaSize, hipMemcpyDeviceToHost);

	double x = h_return[0].InsilicoScore;
	h_LeftMatchedIndex[0] = h_LeftMatchedIndex[0];
	/////////////////////////////////////////////////////////////////

	int SizeOfDataToReturn = 0;
	for (int i = 0; i < candidateProteinsCount; i++)	///CHANGED HERE!!!
	{
		int LeftCount = h_return[i].LeftCount;
		int RightCount = h_return[i].RightCount;
		if (LeftCount > 0 || RightCount > 0)
		{
			(*DataToReturn)->Header = h_return[i].Header;
			(*DataToReturn)->InsilicoScore = h_return[i].InsilicoScore;
			(*DataToReturn)->MatchCounter = h_return[i].MatchCounter;
			(*DataToReturn)->LeftCount = LeftCount;
			(*DataToReturn)->RightCount = RightCount;
			(*DataToReturn)->LeftMatchedIndex = new int[LeftCount];
			(*DataToReturn)->LeftPeakIndex = new int[LeftCount];
			(*DataToReturn)->LeftType = new int[LeftCount];

			int index = i * 5000;
			int ind = 0;
			for (int j = index; j < index + LeftCount; j++) {
				(*DataToReturn)->LeftMatchedIndex[ind] = h_LeftMatchedIndex[j];
				(*DataToReturn)->LeftPeakIndex[ind] = h_LeftPeakIndex[j];
				(*DataToReturn)->LeftType[ind] = h_LeftType[j];
				ind++;
			}
			ind = 0;
			(*DataToReturn)->RightMatchedIndex = new int[RightCount];
			(*DataToReturn)->RightPeakIndex = new int[RightCount];
			(*DataToReturn)->RightType = new int[RightCount];
			for (int j = index; j < index + RightCount; j++) {
				(*DataToReturn)->RightMatchedIndex[ind] = h_RightMatchedIndex[j];
				(*DataToReturn)->RightPeakIndex[ind] = h_RightPeakIndex[j];
				(*DataToReturn)->RightType[ind] = h_RightType[j];
				ind++;
			}
			SizeOfDataToReturn++;
			DataToReturn++;
		}
	}

	return SizeOfDataToReturn;
}
