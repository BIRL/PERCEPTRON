
#include <hip/hip_runtime.h>
//#include <iostream>
//#include <stdio.h>
//#include <cublas.h>
//
//#include <assert.h>
//#include <stdlib.h>
//#include <cuda.h>
//#include <cuda_runtime.h>
//#include <cuda_runtime_api.h>
//#include <device_launch_parameters.h>
//#include  <device_atomic_functions.h>
//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>
//#include <string>
//#include <iterator>
//#include<algorithm>
//#include <chrono>
//#include <math.h>
//#include <vector>
////#include "CudaHeaderFile.cuh"
//using namespace std;
//
//__device__ int index_count = 0;
//
//struct ParametersToCpp
//{
//	double MwTolerance;
//	double NeutralLoss;
//	double SliderValue;
//	double HopThreshhold;
//	int Autotune;
//	int DenovoAllow;
//	int MinimumPstLength;
//	int MaximumPstLength;
//	const char* peptideToleranceUnit;
//	double peptideTolerance;
//};
//
//struct ProteinStructFromCS
//{
//	const char* Header;
//	double* InsilicoMassLeft;
//	double* InsilicoMassRight;
//	double* InsilicoMassLeftAo;
//	double* InsilicoMassLeftBo;
//	double* InsilicoMassLeftAstar;
//	double* InsilicoMassLeftBstar;
//	double* InsilicoMassRightYo;
//	double* InsilicoMassRightYstar;
//	double* InsilicoMassRightZo;
//	double* InsilicoMassRightZoo;
//	int* SizeOfAllInsilicoArrays;
//};
////struct ProteinStructToReturn
////{
////	const char* Header;
////	int MatchCounter;
////	double InsilicoScore;
////	int* LeftMatchedIndex;
////	int* RightMatchedIndex;
////	int* LeftPeakIndex;
////	int* RightPeakIndex;
////	int* LeftType;
////	int* RightType;
////
////};
//struct ProteinStructToReturn
//{
//	int Header;
//	int MatchCounter;
//	double Matches_Score;
//	int LeftMatchedIndex[5000];
//	int RightMatchedIndex[5000];
//	int LeftPeakIndex[5000];
//	int RightPeakIndex[5000];
//	int LeftType[5000];
//	int RightType[5000];
//
//};
//
//typedef struct ToDefineSizeStruct
//{
//	int sizeOfArray;
//}Todefinesizestruct;
//
//int PeptideTolUnitMapping(string pepUnit)
//{
//	int PepUnit;
//	if (pepUnit == "Da")
//		return PepUnit = 1;
//	if (pepUnit == "mmu")
//		return PepUnit = 2;
//	if (pepUnit == "ppm")
//		return PepUnit = 3;
//}
//
//
//
////__device__ void SpectralComparison(double difference, double dev_intensity, int indexPeakList, double peakDifferenceTolerance, int &Consecutive,
////	int &Counter, int &OldConsec, int &OldConsec2, int &ConsecutiveRegion, ProteinStructToReturn *DeviceCandidateProteinReturnPtr, int tid, int indexLeftSide, int Type)
////{
////	double absdifference = fabs(difference);  //Taking Absoulte difference {Doesn't matter}
////	int *acfg;
////	if (absdifference <= peakDifferenceTolerance)
////	{
////		if (Consecutive == OldConsec + 1 && OldConsec == OldConsec2 + 1)
////		{
////			if (Counter == 0)
////			{
////				ConsecutiveRegion = ConsecutiveRegion + 1;
////			}
////			Counter = Counter + 1;
////			DeviceCandidateProteinReturnPtr[tid].Matches_Score = DeviceCandidateProteinReturnPtr[tid].Matches_Score + 1.5;
////			OldConsec2 = OldConsec;
////			OldConsec = Consecutive;
////		}
////		else if (Consecutive == OldConsec && OldConsec == OldConsec2 + 1)
////		{
////			Counter = Counter + 1;
////			Matches_Score = Matches_Score + 1.5;
////		}
////		else
////		{
////			Counter = 0;
////			Matches_Score = Matches_Score + dev_intensity;
////			OldConsec2 = OldConsec;
////			OldConsec = Consecutive;
////		}
////		int insert_ptr = atomicAdd(&index_count, 1);
////		Matched_IndexList[insert_ptr] = indexSide;
////		Peak_IndexList[insert_ptr] = indexPeakList;
////		TypeList[insert_ptr] = Type;
////		MatchCounter = MatchCounter + 1;
////	}
////}
//
//
//
//__global__ void ComputeInsilicoScore(ProteinStructFromCS *h_a, ProteinStructToReturn *DeviceCandidateProteinReturnPtr, 
//	int candidateProteinsCount, ParametersToCpp Parameters,  double *dev_masses, double *dev_intensities, int PeakListCount, int intPepUnit)
//{
//	int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	if (tid < candidateProteinsCount)
//	{
//		ProteinStructFromCS temp = h_a[tid];
//
//
//		int Type;
//		int InsilicoMassLeftCount = temp.SizeOfAllInsilicoArrays[0], InsilicoMassRightCount = temp.SizeOfAllInsilicoArrays[1],
//			InsilicoMassLeftAoCount = temp.SizeOfAllInsilicoArrays[2], InsilicoMassLeftBoCount = temp.SizeOfAllInsilicoArrays[3],
//			InsilicoMassLeftAstarCount = temp.SizeOfAllInsilicoArrays[4], InsilicoMassLeftBstarCount = temp.SizeOfAllInsilicoArrays[5],
//			InsilicoMassRightYoCount = temp.SizeOfAllInsilicoArrays[6], InsilicoMassRightYstarCount = temp.SizeOfAllInsilicoArrays[7],
//			InsilicoMassRightZoCount = temp.SizeOfAllInsilicoArrays[8], InsilicoMassRightZooCount = temp.SizeOfAllInsilicoArrays[9];
//
//
//		int SpecialLeftFragments = temp.SizeOfAllInsilicoArrays[0] + temp.SizeOfAllInsilicoArrays[2] + temp.SizeOfAllInsilicoArrays[3] + 
//			temp.SizeOfAllInsilicoArrays[4] + temp.SizeOfAllInsilicoArrays[5];
//		int SpecialRightFragments  = temp.SizeOfAllInsilicoArrays[1] + temp.SizeOfAllInsilicoArrays[6] + temp.SizeOfAllInsilicoArrays[7] +
//			temp.SizeOfAllInsilicoArrays[8] + temp.SizeOfAllInsilicoArrays[9];
//
//		/*int InsilicoMassLeftCount = temp.SizeOfAllInsilicoArrays[0];
//		int InsilicoMassRightCount = temp.SizeOfAllInsilicoArrays[1];*/
//
//		int Counter = 0; 
//		int OldConsec = -1;      
//		int OldConsec2 = -1;     
//		int ConsecutiveRegion = 0;
//		int IdxL = 0;
//		int IdxR = 0;
//
//		for (int indexPeakList = 1; indexPeakList < PeakListCount; indexPeakList++)
//		{
//			double peakDifferenceTolerance;
//			if (intPepUnit == 1 || intPepUnit == 2)
//				peakDifferenceTolerance = Parameters.peptideTolerance;
//			else if (intPepUnit == 3)
//				peakDifferenceTolerance = (Parameters.peptideTolerance * dev_masses[indexPeakList]) / 1000000;
//			int Consecutive = indexPeakList;
//			for (int indexLeftSide = IdxL; indexLeftSide < InsilicoMassLeftCount; indexLeftSide++)
//			{
//				Type = 0;
//				double difference = dev_masses[indexPeakList] - temp.InsilicoMassLeft[indexLeftSide];
//				SpectralComparison(difference, dev_intensities[indexPeakList], indexPeakList, peakDifferenceTolerance, Consecutive, Counter, OldConsec, OldConsec2, ConsecutiveRegion, DeviceCandidateProteinReturnPtr, tid, indexLeftSide, Type);
//
//				if (SpecialLeftFragments > 0)
//				{
//					if (InsilicoMassLeftAoCount > 0)
//					{
//						Type = 2;
//						difference = dev_masses[indexPeakList] - temp.InsilicoMassLeftAo[indexLeftSide];
//						SpectralComparison(difference, dev_intensities[indexPeakList], indexPeakList, peakDifferenceTolerance, Consecutive, Counter, OldConsec, OldConsec2, ConsecutiveRegion, DeviceCandidateProteinReturnPtr, tid, indexLeftSide, Type);
//					}
//					if (InsilicoMassLeftBoCount > 0)
//					{
//						Type = 3;
//						difference = dev_masses[indexPeakList] - temp.InsilicoMassLeftBo[indexLeftSide];
//						SpectralComparison(difference, dev_intensities[indexPeakList], indexPeakList, peakDifferenceTolerance, Consecutive, Counter, OldConsec, OldConsec2, ConsecutiveRegion, DeviceCandidateProteinReturnPtr, tid, indexLeftSide, Type);
//					}
//					if (InsilicoMassLeftAstarCount > 0)
//					{
//						Type = 4;
//						difference = dev_masses[indexPeakList] - temp.InsilicoMassLeftAstar[indexLeftSide];
//						SpectralComparison(difference, dev_intensities[indexPeakList], indexPeakList, peakDifferenceTolerance, Consecutive, Counter, OldConsec, OldConsec2, ConsecutiveRegion, DeviceCandidateProteinReturnPtr, tid, indexLeftSide, Type);
//					}
//					if (InsilicoMassLeftBstarCount > 0)
//					{
//						Type = 5;
//						difference = dev_masses[indexPeakList] - temp.InsilicoMassLeftBstar[indexLeftSide];
//						SpectralComparison(difference, dev_intensities[indexPeakList], indexPeakList, peakDifferenceTolerance, Consecutive, Counter, OldConsec, OldConsec2, ConsecutiveRegion, DeviceCandidateProteinReturnPtr, tid, indexLeftSide, Type);
//					}
//				}
//				if (difference < -peakDifferenceTolerance && indexLeftSide > 0)  // Updated 20200917   -- Changed from 1 to 0
//				{
//					IdxL = indexLeftSide - 1;
//					break;
//				}
//			}
//			
//			
//			for (int indexRightSide = IdxR; indexRightSide < InsilicoMassLeftCount; indexRightSide++)
//			{
//				Type = 6;
//				double difference = dev_masses[indexPeakList] - temp.InsilicoMassRight[indexRightSide];
//				SpectralComparison(difference, dev_intensities[indexPeakList], indexPeakList, peakDifferenceTolerance, Consecutive, Counter, OldConsec, OldConsec2, ConsecutiveRegion, DeviceCandidateProteinReturnPtr, tid, indexRightSide, Type);
//
//				if (SpecialLeftFragments > 0)
//				{
//					if (InsilicoMassRightYoCount > 0)
//					{
//						Type = 7;
//						difference = dev_masses[indexPeakList] - temp.InsilicoMassRightYo[indexRightSide];
//						SpectralComparison(difference, dev_intensities[indexPeakList], indexPeakList, peakDifferenceTolerance, Consecutive, Counter, OldConsec, OldConsec2, ConsecutiveRegion, DeviceCandidateProteinReturnPtr, tid, indexRightSide, Type);
//					}
//					if (InsilicoMassRightYstarCount > 0)
//					{
//						Type = 8;
//						difference = dev_masses[indexPeakList] - temp.InsilicoMassRightYstar[indexRightSide];
//						SpectralComparison(difference, dev_intensities[indexPeakList], indexPeakList, peakDifferenceTolerance, Consecutive, Counter, OldConsec, OldConsec2, ConsecutiveRegion, DeviceCandidateProteinReturnPtr, tid, indexRightSide, Type);
//					}
//					if (InsilicoMassRightZoCount > 0)
//					{
//						Type = 9;
//						difference = dev_masses[indexPeakList] - temp.InsilicoMassRightZo[indexRightSide];
//						SpectralComparison(difference, dev_intensities[indexPeakList], indexPeakList, peakDifferenceTolerance, Consecutive, Counter, OldConsec, OldConsec2, ConsecutiveRegion, DeviceCandidateProteinReturnPtr, tid, indexRightSide, Type);
//					}
//					if (InsilicoMassRightZooCount > 0)
//					{
//						Type = 10;
//						difference = dev_masses[indexPeakList] - temp.InsilicoMassRightZoo[indexRightSide];
//						SpectralComparison(difference, dev_intensities[indexPeakList], indexPeakList, peakDifferenceTolerance, Consecutive, Counter, OldConsec, OldConsec2, ConsecutiveRegion, DeviceCandidateProteinReturnPtr, tid, indexRightSide, Type);
//					}
//				}
//				if (difference < -peakDifferenceTolerance && indexRightSide > 0)  // Updated 20200917   -- Changed from 1 to 0
//				{
//					IdxR = indexRightSide - 1;
//					break;
//				}
//			}
//		}
//
//		
//
//		//int s = temp.SizeOfAllInsilicoArrays[0];
//
//		//int* Left;
//		//int* Right;
//
//		//int IdxL = 0;
//		//int IdxR = 0;
//		//int* LeftType;
//		//int* RightType;
//
//
//
//
//		h_a[tid].InsilicoMassLeft[0] = 5;
//	}
//}
//
//extern "C" __declspec(dllexport) void __cdecl
//insilicospectralcomparisongpu(ParametersToCpp Parameters, ProteinStructFromCS **candidateProteins, double *PeakListMasses, 
//	double *PeakListIntensities, int PeakListCount, int candidateProteinsCount)
//{
//	ProteinStructFromCS *CandidateProteinsToCuda = new ProteinStructFromCS;
//	ProteinStructFromCS *dev_CandidateProteinsToCuda;
//
//	thrust::host_vector<ProteinStructFromCS> host_CandidateProteinsToCuda;
//	ProteinStructFromCS *h_a = new ProteinStructFromCS[candidateProteinsCount];
//	ProteinStructFromCS *d_a;
//	int *SizeOfAllInsilicoArrays = new int[10];
//	
//	//
//	int SizeOfLeftIonsTotal = 0;
//	int *SizeOfIndividualLeftIons = new int[candidateProteinsCount];
//	//
//	int SizeOfRightIonsTotal = 0;
//	int *SizeOfIndividualRightIons = new int[candidateProteinsCount];
//
//	int SizeOfLeftAoIonsTotal = 0;
//	int *SizeOfIndividualLeftAoIons = new int[candidateProteinsCount];
//
//	int SizeOfLeftBoIonsTotal = 0;
//	int *SizeOfIndividualLeftBoIons = new int[candidateProteinsCount];
//
//	int SizeOfLeftAstarIonsTotal = 0;
//	int *SizeOfIndividualLeftAstarIons = new int[candidateProteinsCount];
//
//
//	int SizeOfLeftBstarIonsTotal = 0;
//	int *SizeOfIndividualLeftBstarIons = new int[candidateProteinsCount];
//
//
//	int SizeOfRightYoIonsTotal = 0;
//	int *SizeOfIndividualRightYoIons = new int[candidateProteinsCount];
//
//	int SizeOfRightYstarIonsTotal = 0;
//	int *SizeOfIndividualRightYstarIons = new int[candidateProteinsCount];
//
//	int SizeOfRightZoIonsTotal = 0;
//	int *SizeOfIndividualRightZoIons = new int[candidateProteinsCount];
//
//	int SizeOfRightZooIonsTotal = 0;
//	int *SizeOfIndividualRightZooIons = new int[candidateProteinsCount];
//	
//	
//	
//	
//	//double *h_arr;	//HERE	double **h_arr = new double*[4];
//	/*double *d_arr;*/
//	for (int i = 0; i < candidateProteinsCount; i++)
//	{		
//		SizeOfAllInsilicoArrays[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[0];
//		
//		
//		CandidateProteinsToCuda[i] = (**candidateProteins);
//		SizeOfIndividualLeftIons[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[0]; //SizeOfAllInsilicoArrays[0];
//		SizeOfLeftIonsTotal = SizeOfLeftIonsTotal + (*candidateProteins)->SizeOfAllInsilicoArrays[0]; //SizeOfAllInsilicoArrays[0];
//		//
//
//		SizeOfIndividualRightIons[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[1]; //SizeOfAllInsilicoArrays[0];
//		SizeOfRightIonsTotal = SizeOfRightIonsTotal + (*candidateProteins)->SizeOfAllInsilicoArrays[1]; //SizeOfAllInsilicoArrays[0];
//
//		SizeOfIndividualLeftAoIons[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[2]; //SizeOfAllInsilicoArrays[0];
//		SizeOfLeftAoIonsTotal = SizeOfLeftAoIonsTotal + (*candidateProteins)->SizeOfAllInsilicoArrays[2]; //SizeOfAllInsilicoArrays[0];
//
//		SizeOfIndividualLeftBoIons[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[3]; //SizeOfAllInsilicoArrays[0];
//		SizeOfLeftBoIonsTotal = SizeOfLeftBoIonsTotal + (*candidateProteins)->SizeOfAllInsilicoArrays[3]; //SizeOfAllInsilicoArrays[0];
//
//		SizeOfIndividualLeftAstarIons[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[4]; //SizeOfAllInsilicoArrays[0];
//		SizeOfLeftAstarIonsTotal = SizeOfLeftAstarIonsTotal + (*candidateProteins)->SizeOfAllInsilicoArrays[4]; //SizeOfAllInsilicoArrays[0];
//
//
//
//
//		SizeOfIndividualLeftBstarIons[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[5]; //SizeOfAllInsilicoArrays[0];
//		SizeOfLeftBstarIonsTotal = SizeOfLeftBstarIonsTotal + (*candidateProteins)->SizeOfAllInsilicoArrays[5]; //SizeOfAllInsilicoArrays[0];
//
//		SizeOfIndividualRightYoIons[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[6]; //SizeOfAllInsilicoArrays[0];
//		SizeOfRightYoIonsTotal = SizeOfRightYoIonsTotal + (*candidateProteins)->SizeOfAllInsilicoArrays[6]; //SizeOfAllInsilicoArrays[0];
//
//		SizeOfIndividualRightYstarIons[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[7]; //SizeOfAllInsilicoArrays[0];
//		SizeOfRightYstarIonsTotal = SizeOfRightYstarIonsTotal + (*candidateProteins)->SizeOfAllInsilicoArrays[7]; //SizeOfAllInsilicoArrays[0];
//
//		SizeOfIndividualRightZoIons[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[8]; //SizeOfAllInsilicoArrays[0];
//		SizeOfRightZoIonsTotal = SizeOfRightZoIonsTotal + (*candidateProteins)->SizeOfAllInsilicoArrays[8]; //SizeOfAllInsilicoArrays[0];
//
//
//		SizeOfIndividualRightZooIons[i] = (*candidateProteins)->SizeOfAllInsilicoArrays[9]; //SizeOfAllInsilicoArrays[0];
//		SizeOfRightZooIonsTotal = SizeOfRightZooIonsTotal + (*candidateProteins)->SizeOfAllInsilicoArrays[9]; //SizeOfAllInsilicoArrays[0];
//
//
//		host_CandidateProteinsToCuda.push_back(**candidateProteins);
//		
//		candidateProteins++;
//	}
//	//CandidateProteinsToCuda[0] = CandidateProteinsToCuda[0];
//	//CandidateProteinsToCuda[1] = CandidateProteinsToCuda[1];
//	//CandidateProteinsToCuda[2] = CandidateProteinsToCuda[2];
//	//CandidateProteinsToCuda[3] = CandidateProteinsToCuda[3];
//	////CandidateProteinsToCuda[0].SizeOfAllInsilicoArrays[0];
//	double* h_arr = new double[SizeOfLeftIonsTotal];	//HERE	h_arr = new double[SizeOfAllInsilicoArrays[0]];
//	//
//
//	double* h_arr_Right = new double[SizeOfRightIonsTotal];
//	double* h_arr_LeftAo = new double[SizeOfLeftAoIonsTotal];
//	double* h_arr_LeftBo = new double[SizeOfLeftBoIonsTotal];
//	double* h_arr_LeftAstar = new double[SizeOfLeftAstarIonsTotal];
//	double* h_arr_LeftBstar = new double[SizeOfLeftBstarIonsTotal];
//	double* h_arr_RightYo = new double[SizeOfRightYoIonsTotal];
//	double* h_arr_RightYstar = new double[SizeOfRightYstarIonsTotal];
//	double* h_arr_RightZo = new double[SizeOfRightZoIonsTotal];
//	double* h_arr_RightZoo = new double[SizeOfRightZooIonsTotal];
//
//	int k = 0, kRight = 0, kLeftAo = 0, kLeftBo = 0, kLeftAstar = 0, kLeftBstar = 0,
//		kRightYo = 0, kRightYstar = 0, kRightZo = 0, kRightZoo = 0;
//
//
//	for (int i = 0; i < candidateProteinsCount; i++)
//	{
//		///Left
//		int InnerCount = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[0];
//		for (int j = 0; j < InnerCount; j++)
//		{
//			h_arr[k] = host_CandidateProteinsToCuda[i].InsilicoMassLeft[j];
//			k++;
//		}
//		////
//
//		int InnerCountRight = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[1];
//		for (int j = 0; j < InnerCountRight; j++)
//		{
//			h_arr_Right[kRight] = host_CandidateProteinsToCuda[i].InsilicoMassLeft[j];
//			kRight++;
//		}
//
//
//		int InnerCountLeftAo = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[2];
//		for (int j = 0; j < InnerCountLeftAo; j++)
//		{
//			h_arr_LeftAo[kLeftAo] = host_CandidateProteinsToCuda[i].InsilicoMassLeft[j];
//			kLeftAo++;
//		}
//
//
//		int InnerCountLeftBo = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[3];
//		for (int j = 0; j < InnerCountLeftBo; j++)
//		{
//			h_arr_LeftBo[kLeftBo] = host_CandidateProteinsToCuda[i].InsilicoMassLeft[j];
//			kLeftBo++;
//		}
//
//
//		int InnerCountLeftAstar = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[4];
//		for (int j = 0; j < InnerCountLeftAstar; j++)
//		{
//			h_arr_LeftAstar[kLeftAstar] = host_CandidateProteinsToCuda[i].InsilicoMassLeft[j];
//			kLeftAstar++;
//		}
//
//
//		int InnerCountLeftBstar = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[5];
//		for (int j = 0; j < InnerCountLeftBstar; j++)
//		{
//			h_arr_LeftBstar[kLeftBstar] = host_CandidateProteinsToCuda[i].InsilicoMassLeft[j];
//			kLeftBstar++;
//		}
//
//		int InnerCountRightYo = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[6];
//		for (int j = 0; j < InnerCountRightYo; j++)
//		{
//			h_arr_RightYo[kRightYo] = host_CandidateProteinsToCuda[i].InsilicoMassLeft[j];
//			kRightYo++;
//		}
//
//
//		int InnerCountRightYstar = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[7];
//		for (int j = 0; j < InnerCountRightYstar; j++)
//		{
//			h_arr_RightYstar[kRightYstar] = host_CandidateProteinsToCuda[i].InsilicoMassLeft[j];
//			kRightYstar++;
//		}
//
//
//		int InnerCountRightZo = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[8];
//		for (int j = 0; j < InnerCountRightZo; j++)
//		{
//			h_arr_RightZo[kRightZo] = host_CandidateProteinsToCuda[i].InsilicoMassLeft[j];
//			kRightZo++;
//		}
//
//
//		int InnerCountRightZoo = CandidateProteinsToCuda[i].SizeOfAllInsilicoArrays[9];
//		for (int j = 0; j < InnerCountRightZoo; j++)
//		{
//			h_arr_RightZoo[kRightZoo] = host_CandidateProteinsToCuda[i].InsilicoMassLeft[j];
//			kRightZoo++;
//		}
//	}
//	
//	double *d_arr, *d_arrRight, *d_arrLeftAo, *d_arrLeftBo, *d_arrLeftAstar, *d_arrLeftBstar,
//		*d_arrRightYo, *d_arrRightYstar, *d_arrRightZo, *d_arrRightZoo;
//
//
//	cudaMalloc((void**) &(d_arr), sizeof(double)*SizeOfLeftIonsTotal);
//	cudaMemcpy(d_arr, h_arr, sizeof(double)*SizeOfLeftIonsTotal, cudaMemcpyHostToDevice);
//	//
//
//
//	cudaMalloc((void**) &(d_arrRight), sizeof(double)*SizeOfRightIonsTotal);
//	cudaMemcpy(d_arrRight, h_arr_Right, sizeof(double)*SizeOfRightIonsTotal, cudaMemcpyHostToDevice);
//
//
//	cudaMalloc((void**) &(d_arrLeftAo), sizeof(double)*SizeOfLeftAoIonsTotal);
//	cudaMemcpy(d_arrLeftAo, h_arr_LeftAo, sizeof(double)*SizeOfLeftAoIonsTotal, cudaMemcpyHostToDevice);
//
//	cudaMalloc((void**) &(d_arrLeftBo), sizeof(double)*SizeOfLeftBoIonsTotal);
//	cudaMemcpy(d_arrLeftBo, h_arr_LeftBo, sizeof(double)*SizeOfLeftBoIonsTotal, cudaMemcpyHostToDevice);
//
//	cudaMalloc((void**) &(d_arrLeftAstar), sizeof(double)*SizeOfLeftAstarIonsTotal);
//	cudaMemcpy(d_arrLeftAstar, h_arr_LeftAstar, sizeof(double)*SizeOfLeftAstarIonsTotal, cudaMemcpyHostToDevice);
//
//	cudaMalloc((void**) &(d_arrLeftBstar), sizeof(double)*SizeOfLeftBstarIonsTotal);
//	cudaMemcpy(d_arrLeftBstar, h_arr_LeftBstar, sizeof(double)*SizeOfLeftBstarIonsTotal, cudaMemcpyHostToDevice);
//
//	cudaMalloc((void**) &(d_arrRightYo), sizeof(double)*SizeOfRightYoIonsTotal);
//	cudaMemcpy(d_arrRightYo, h_arr_RightYo, sizeof(double)*SizeOfRightYoIonsTotal, cudaMemcpyHostToDevice);
//
//	cudaMalloc((void**) &(d_arrRightYstar), sizeof(double)*SizeOfRightYstarIonsTotal);
//	cudaMemcpy(d_arrRightYstar, h_arr_RightYstar, sizeof(double)*SizeOfRightYstarIonsTotal, cudaMemcpyHostToDevice);
//
//	cudaMalloc((void**) &(d_arrRightZo), sizeof(double)*SizeOfRightZoIonsTotal);
//	cudaMemcpy(d_arrRightZo, h_arr_RightZo, sizeof(double)*SizeOfRightZoIonsTotal, cudaMemcpyHostToDevice);
//
//	cudaMalloc((void**) &(d_arrRightZoo), sizeof(double)*SizeOfRightZooIonsTotal);
//	cudaMemcpy(d_arrRightZoo, h_arr_RightZoo, sizeof(double)*SizeOfRightZooIonsTotal, cudaMemcpyHostToDevice);
//
//
//	int PrevLeftIons = 0, PrevRightIons = 0, PrevLeftAoIons = 0, PrevLeftBoIons = 0,
//		PrevLeftAstarIons = 0, PrevLeftBstarIons = 0,
//		PrevRightYoIons = 0, PrevRightYstarIons = 0, PrevRightZoIons = 0, PrevRightZooIons = 0;
//		
//	for (int i = 0; i < candidateProteinsCount; i++)
//	{
//		h_a[i].InsilicoMassLeft = d_arr + (PrevLeftIons);
//		PrevLeftIons = PrevLeftIons + SizeOfIndividualLeftIons[i];
//		///
//
//		h_a[i].InsilicoMassRight = d_arrRight + (PrevRightIons);
//		PrevRightIons = PrevRightIons + SizeOfIndividualRightIons[i];
//
//		h_a[i].InsilicoMassLeftAo = d_arrLeftAo + (PrevLeftAoIons);
//		PrevLeftAoIons = PrevLeftAoIons + SizeOfIndividualLeftAoIons[i];
//
//		h_a[i].InsilicoMassLeftBo = d_arrLeftBo + (PrevLeftBoIons);
//		PrevLeftBoIons = PrevLeftBoIons + SizeOfIndividualLeftBoIons[i];
//
//		h_a[i].InsilicoMassLeftAstar = d_arrLeftAstar + (PrevLeftAstarIons);
//		PrevLeftAstarIons = PrevLeftAstarIons + SizeOfIndividualLeftAstarIons[i];
//
//		h_a[i].InsilicoMassLeftBstar = d_arrLeftBstar + (PrevLeftBstarIons);
//		PrevLeftBstarIons = PrevLeftBstarIons + SizeOfIndividualLeftBstarIons[i];
//
//		h_a[i].InsilicoMassRightYo = d_arrRightYo + (PrevRightYoIons);
//		PrevRightYoIons = PrevRightYoIons + SizeOfIndividualRightYoIons[i];
//
//		h_a[i].InsilicoMassRightYstar = d_arrRightYstar + (PrevRightYstarIons);
//		PrevRightYstarIons = PrevRightYstarIons + SizeOfIndividualRightYstarIons[i];
//
//		h_a[i].InsilicoMassRightZo = d_arrRightZo + (PrevRightZoIons);
//		PrevRightZoIons = PrevRightZoIons + SizeOfIndividualRightZoIons[i];
//
//		h_a[i].InsilicoMassRightZoo = d_arrRightZoo + (PrevRightZooIons);
//		PrevRightZooIons = PrevRightZooIons + SizeOfIndividualRightZooIons[i];
//
//	}
//
//	cudaMalloc((void**) &(d_a), sizeof(ProteinStructFromCS)*candidateProteinsCount);
//	cudaMemcpy(d_a, h_a, sizeof(ProteinStructFromCS)*candidateProteinsCount, cudaMemcpyHostToDevice);
//	
//	int NumOfThreadsToLaunch = 256;
//	int NoOfBlocks = NumOfThreadsToLaunch/candidateProteinsCount;
//	
//	std::string stringPepUnit = Parameters.peptideToleranceUnit;
//	int intPepUnit = PeptideTolUnitMapping(stringPepUnit);
//
//	double *dev_masses, *dev_intensities;
//	cudaMalloc((void**)&dev_masses, sizeof(double) * PeakListCount);
//	cudaMalloc((void**)&dev_intensities, sizeof(double) * PeakListCount);
//	cudaMemcpy(dev_masses, PeakListMasses, PeakListCount * sizeof(double), cudaMemcpyHostToDevice);
//	cudaMemcpy(dev_intensities, PeakListIntensities, PeakListCount * sizeof(double), cudaMemcpyHostToDevice);
//
//	thrust::device_vector<ProteinStructToReturn> DeviceCandidateProteinReturn(candidateProteinsCount);
//	ProteinStructToReturn *DeviceCandidateProteinReturnPtr = thrust::raw_pointer_cast(DeviceCandidateProteinReturn.data());
//
//	ComputeInsilicoScore << <NoOfBlocks, NumOfThreadsToLaunch >> > (d_a, DeviceCandidateProteinReturnPtr);
//	cudaMemcpy(h_a, d_a, sizeof(ProteinStructFromCS)*candidateProteinsCount, cudaMemcpyDeviceToHost);
//
//	thrust::host_vector<ProteinStructToReturn> HostCandidateProteinReturn = DeviceCandidateProteinReturn;
//	
//
//
//
//
//
//
//	//h_a[0].InsilicoMassLeft[0] = h_a[0].InsilicoMassLeft[0];
//	h_arr[0] = h_arr[0];
//	/*h_a[0].InsilicoMassLeft = h_arr;
//	h_a[1].InsilicoMassLeft = h_arr + SizeOfAllInsilicoArrays[0];*/
//	vector<double> u;
//	for (int i = 0; i < SizeOfLeftIonsTotal; i++)
//	{
//		u.push_back(h_arr[i]);
//	}
//	u[0] = u[0];
//}
//
//
////
////cudaMemcpy(h_arr, d_arr, sizeof(double) * SizeOfLeftIonsTotal, cudaMemcpyDeviceToHost);
////cudaMemcpy(h_arr_Right, d_arrRight, sizeof(double) * SizeOfRightIonsTotal, cudaMemcpyDeviceToHost);
////cudaMemcpy(h_arr_LeftAo, d_arrLeftAo, sizeof(double) * SizeOfLeftAoIonsTotal, cudaMemcpyDeviceToHost);
////cudaMemcpy(h_arr_LeftBo, d_arrLeftBo, sizeof(double) * SizeOfLeftBoIonsTotal, cudaMemcpyDeviceToHost);
////cudaMemcpy(h_arr_LeftAstar, d_arrLeftAstar, sizeof(double) * SizeOfLeftAstarIonsTotal, cudaMemcpyDeviceToHost);
////cudaMemcpy(h_arr_LeftBstar, d_arrLeftBstar, sizeof(double) * SizeOfLeftBstarIonsTotal, cudaMemcpyDeviceToHost);
////cudaMemcpy(h_arr_RightYo, d_arrRightYo, sizeof(double) * SizeOfRightYoIonsTotal, cudaMemcpyDeviceToHost);
////cudaMemcpy(h_arr_RightYstar, d_arrRightYstar, sizeof(double) * SizeOfRightYstarIonsTotal, cudaMemcpyDeviceToHost);
////cudaMemcpy(h_arr_RightZo, d_arrRightZo, sizeof(double) * SizeOfRightZoIonsTotal, cudaMemcpyDeviceToHost);
////cudaMemcpy(h_arr_RightZoo, d_arrRightZoo, sizeof(double) * SizeOfRightZooIonsTotal, cudaMemcpyDeviceToHost);