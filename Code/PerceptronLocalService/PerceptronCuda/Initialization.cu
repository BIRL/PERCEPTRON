#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>


using namespace std;

extern "C" __declspec(dllexport) void __cdecl MainInitializer()
{
	int *GpuIsInitializing;
	hipMalloc((void**)&GpuIsInitializing, sizeof(int) * 10);
}

