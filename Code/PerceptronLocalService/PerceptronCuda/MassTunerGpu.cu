#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hipblas.h>
#include <vector>
#include <assert.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include  <device_atomic_functions.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <string>
#include <iterator>
#include<algorithm>
#include <chrono>

#define SIZE	1024 
using namespace std;

//	--- Updated: 20210223 ---
__device__ int dev_data[50];
__device__ int dev_count = 0;
__device__ int dev_wind_count = 0;

typedef struct _WindowCapturedElementsStruct
{
	double TunedMass;
	int elementCount;
} windowcapturedelementsstruct;

typedef struct _ShortlistedMassSumsAndIntensities
{
	double massSum;
	double AvgIntensity;
	bool operator() (_ShortlistedMassSumsAndIntensities i, _ShortlistedMassSumsAndIntensities j) { return (i.massSum < j.massSum); }
} ShortlistedMassSumsAndIntensities;

__device__ void my_push_back(double *dev_PeakListMassesSum, double *dev_PeakListIntensitiesAverage, double summationOfMasses, double averageOfIntensities)
{
	int insert_pt = atomicAdd(&dev_count, 1);
	dev_PeakListMassesSum[insert_pt] = summationOfMasses;
	dev_PeakListIntensitiesAverage[insert_pt] = averageOfIntensities;
	return;
}

__global__ void vectorAdd(double *raw_ptr_masses, double *raw_ptr_intensities, double *dev_PeakListMassesSum, double *dev_PeakListIntensitiesAverage, double MwTolerance, double NeutralLoss, int N) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < N)
	{
		for (int i = tid + 1; i < N; i++)
		{
			double summationOfMasses = raw_ptr_masses[tid] + raw_ptr_masses[i] + NeutralLoss;
			double averageOfIntensities = (raw_ptr_intensities[tid] + raw_ptr_intensities[i]) / 2;
			my_push_back(dev_PeakListMassesSum, dev_PeakListIntensitiesAverage, summationOfMasses, averageOfIntensities);
		}
	}
	else
		return;
}

__device__ void window_push_back(_WindowCapturedElementsStruct *windowcapturedelements, double a, int b)
{
	int insert_ptr = atomicAdd(&dev_wind_count, 1);
	windowcapturedelements[insert_ptr].TunedMass = a;
	windowcapturedelements[insert_ptr].elementCount = b;
	return;
}

__global__ void WindowLaunchKernel(int NumOfThreadsToLaunch, double minSum, double maxSum, _ShortlistedMassSumsAndIntensities *shortListedData, int sizeOfShortlistedData, _WindowCapturedElementsStruct *windowcapturedelements, double SliderValue)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < NumOfThreadsToLaunch)
	{
		double WindowStart = minSum + (tid * SliderValue);
		double WindowEnd = WindowStart + 1.00727647;
		double sumoftunedmassesandintensities = 0;
		double sumoftunedintensities = 0;
		int Count = 0;
		for (int i = 0; i < sizeOfShortlistedData; i++)
		{
			if (WindowStart <= shortListedData[i].massSum && shortListedData[i].massSum < WindowEnd)//#DISCUSSION
			{
				double data = shortListedData[i].massSum * shortListedData[i].AvgIntensity;
				sumoftunedmassesandintensities = sumoftunedmassesandintensities + data;
				sumoftunedintensities = sumoftunedintensities + shortListedData[i].AvgIntensity;
				Count = Count + 1;
			}
			else if (shortListedData[i].massSum >= WindowEnd)
			{
				break;
			}
		}
		double TunedMass = sumoftunedmassesandintensities / sumoftunedintensities;
		int elementCount = Count;
		window_push_back(windowcapturedelements, TunedMass, elementCount);
	}
}

extern "C" __declspec(dllexport) double __cdecl
wholeproteinmasstuner(double PeakListMasses[], double PeakListIntensities[], int PeakListLength, double MwTolerance, double NeutralLoss, double Slider_Value)
{
	double WholeProteinMass = PeakListMasses[0];

	const int N = PeakListLength;
	const int zN = (floor(PeakListLength*PeakListLength) / 2) - (floor(PeakListLength / 2));
	double *dev_masses, *dev_intensities, *dev_PeakListMassesSum, *dev_PeakListIntensitiesAverage;

	hipMalloc((void**)&dev_masses, sizeof(double)*N);
	hipMalloc((void**)&dev_intensities, sizeof(double)*N);

	thrust::device_vector<double> DevicePeakListMassesSum(zN);
	double *raw_ptr00 = thrust::raw_pointer_cast(DevicePeakListMassesSum.data());

	thrust::device_vector<double> DevicePeakListAvgIntensities(zN);
	double *raw_ptr01 = thrust::raw_pointer_cast(DevicePeakListAvgIntensities.data());

	hipMemcpy(dev_masses, PeakListMasses, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_intensities, PeakListIntensities, N * sizeof(double), hipMemcpyHostToDevice);

	int THREADS = 256;
	int BLOCKS = (N + THREADS - 1);
	vectorAdd << < BLOCKS, THREADS >> > (dev_masses, dev_intensities, raw_ptr00, raw_ptr01, MwTolerance, NeutralLoss, N);

	thrust::host_vector<double> PeakListMassesSum = DevicePeakListMassesSum;
	thrust::host_vector<double> PeakListIntensitiesAverage = DevicePeakListAvgIntensities;
	thrust::host_vector<_ShortlistedMassSumsAndIntensities> shortlistedMassSumAndIntensities;
	for (int i = 0; i < zN; i++)
	{
		double a = PeakListMassesSum[i];
		if (PeakListMasses[0] - MwTolerance <= PeakListMassesSum[i] && PeakListMassesSum[i] <= PeakListMasses[0] + MwTolerance)
		{
			_ShortlistedMassSumsAndIntensities data;
			data.massSum = PeakListMassesSum[i];
			data.AvgIntensity = PeakListIntensitiesAverage[i];
			shortlistedMassSumAndIntensities.push_back(data);
		}
	}

	std::sort(shortlistedMassSumAndIntensities.begin(), shortlistedMassSumAndIntensities.end(),
		[](const _ShortlistedMassSumsAndIntensities &mass, const _ShortlistedMassSumsAndIntensities &mass2)
	{ return (mass.massSum < mass2.massSum); });

	int x = 0;
	double minSum = shortlistedMassSumAndIntensities[0].massSum;
	double maxSum = shortlistedMassSumAndIntensities[shortlistedMassSumAndIntensities.size() - 1].massSum;

	int sizeOfShortlistedData = shortlistedMassSumAndIntensities.size();
	double SliderValue = (WholeProteinMass * Slider_Value) / (pow(10.0, 6.0));
	int NumOfThreadsToLaunch = floor((maxSum - minSum) * (1 / SliderValue));

	thrust::device_vector<_ShortlistedMassSumsAndIntensities> device_shortlistedMassSumAndIntensities = shortlistedMassSumAndIntensities;
	_ShortlistedMassSumsAndIntensities *raw_ptr = thrust::raw_pointer_cast(device_shortlistedMassSumAndIntensities.data());

	thrust::device_vector<_WindowCapturedElementsStruct> device_windowcapturedelements(NumOfThreadsToLaunch);
	_WindowCapturedElementsStruct *raw_ptr2 = thrust::raw_pointer_cast(device_windowcapturedelements.data());

	int THREADS2 = 256;
	int BLOCKS2 = (NumOfThreadsToLaunch / THREADS + 1);

	WindowLaunchKernel << <BLOCKS2, THREADS2 >> > (NumOfThreadsToLaunch, minSum, maxSum, raw_ptr, sizeOfShortlistedData, raw_ptr2, SliderValue);

	thrust::host_vector<_WindowCapturedElementsStruct> host_windowcapturedelements = device_windowcapturedelements;

	double TunedMass = 0;
	int oldElementCount = 0;

	for (int x = 0; x < NumOfThreadsToLaunch; x++)
	{
		if (oldElementCount < host_windowcapturedelements[x].elementCount)
		{
			oldElementCount = host_windowcapturedelements[x].elementCount;
			TunedMass = host_windowcapturedelements[x].TunedMass;
		}
		else if (oldElementCount == host_windowcapturedelements[x].elementCount)
		{
			if (abs(TunedMass - WholeProteinMass) >= abs(host_windowcapturedelements[x].TunedMass - WholeProteinMass))
			{
				TunedMass = host_windowcapturedelements[x].TunedMass;
			}
		}
	}
	hipDeviceSynchronize();
	return TunedMass;
}
