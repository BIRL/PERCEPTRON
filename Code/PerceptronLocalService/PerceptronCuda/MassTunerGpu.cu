#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hipblas.h>
#include <vector>
#include <assert.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include  <device_atomic_functions.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <string>
#include <iterator>
#include<algorithm>
#include <chrono>
#include <math.h>

#define SIZE	1024 
using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

//	--- Updated: 20210223 ---
__device__ int dev_data[50];
__device__ int dev_count = 0;
__device__ int dev_wind_count = 0;
__device__ int dev_pst_count = 0;
__device__ int multipleLengthPstCounter = 0;
__device__ int charArrayCounter = 0;

typedef struct _WindowCapturedElementsStruct
{
	double TunedMass;
	int elementCount;
} windowcapturedelementsstruct;

typedef struct _ShortlistedMassSumsAndIntensities
{
	double massSum;
	double AvgIntensity;
	bool operator() (_ShortlistedMassSumsAndIntensities i, _ShortlistedMassSumsAndIntensities j) { return (i.massSum < j.massSum); }
} ShortlistedMassSumsAndIntensities;

typedef struct _DataForPsts
{
	int startIndex;
	int endIndex;
	double startIndexMass;
	double endIndexMass;
	double massDifferenceBetweenPeaks;
	char AminoAcidSymbol;
	double TagError;
	double averageIntensity;
} dataforpsts;

typedef struct _PeptideSequenceTags
{
	char PstTag[8];
	int PstTagLength;
	double PstErrorScore;
	double PstFrequency;
	int startIndex;
	int endIndex;
} peptidesequencetags;

__device__ void my_push_back(double *dev_PeakListMassesSum, double *dev_PeakListIntensitiesAverage, double summationOfMasses, double averageOfIntensities)
{
	int insert_pt = atomicAdd(&dev_count, 1);
	dev_PeakListMassesSum[insert_pt] = summationOfMasses;
	dev_PeakListIntensitiesAverage[insert_pt] = averageOfIntensities;
	return;
}

__device__ void PST_push_back(_DataForPsts *SingleLengthPSTs_ptr, int tid, int i, double StartIndMass, double EndIndMass, double differenceOfMasses, char aminoAcidSymbol, double TagError, double averageOfIntensities)
{
	int insert_ptr = atomicAdd(&dev_pst_count, 1);
	SingleLengthPSTs_ptr[insert_ptr].startIndex = tid;
	SingleLengthPSTs_ptr[insert_ptr].endIndex = i;
	SingleLengthPSTs_ptr[insert_ptr].startIndexMass = StartIndMass;
	SingleLengthPSTs_ptr[insert_ptr].endIndexMass = EndIndMass;
	SingleLengthPSTs_ptr[insert_ptr].massDifferenceBetweenPeaks = differenceOfMasses;
	SingleLengthPSTs_ptr[insert_ptr].AminoAcidSymbol = aminoAcidSymbol;
	SingleLengthPSTs_ptr[insert_ptr].TagError = TagError;
	SingleLengthPSTs_ptr[insert_ptr].averageIntensity = averageOfIntensities;
	return;
}

__global__ void vectorAdd(double *raw_ptr_masses, double *raw_ptr_intensities, double *dev_PeakListMassesSum, double *dev_PeakListIntensitiesAverage, _DataForPsts *SingleLengthPSTs_ptr, double *dev_aminoAcidMassesList, char *dev_aminoAcidSymbolList, double MwTolerance, double NeutralLoss, double HopThreshold, int N) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < N)
	{
		for (int i = tid + 1; i < N; i++)
		{
			double summationOfMasses = raw_ptr_masses[tid] + raw_ptr_masses[i] + NeutralLoss;
			double averageOfIntensities = (raw_ptr_intensities[tid] + raw_ptr_intensities[i]) / 2;
			my_push_back(dev_PeakListMassesSum, dev_PeakListIntensitiesAverage, summationOfMasses, averageOfIntensities);
			double differenceOfMasses = fabs(raw_ptr_masses[tid] - raw_ptr_masses[i]);
			for (int j = 0; j < 21; j++)
			{
				double TagError = fabs(dev_aminoAcidMassesList[j] - differenceOfMasses);
				if (TagError <= HopThreshold)
				{
					PST_push_back(SingleLengthPSTs_ptr, tid, i, raw_ptr_masses[tid], raw_ptr_masses[i], differenceOfMasses, dev_aminoAcidSymbolList[j], TagError, averageOfIntensities);
				}
			}
			
		}
	}
	else
		return;
}

__device__ char * my_strcpy(char *dest, const char *src) {
	int i = 0;
	do {
		dest[i] = src[i];
	} while (src[i++] != 0);
	return dest;
}

//__device__ void GeneratingMultipleLengthPstsInner(int HomePeak, char HomePeakAA, _PeptideSequenceTags *MultipleLengthPst_ptr, _DataForPsts *SingleLengthPSTs, char *tempPstArray, int IndexToInsertAA, int N)
//{
//	int i;
//	for (i=0; i<N; i++)
//	{
//		int HopPeak = SingleLengthPSTs[i].startIndex;
//		char HopPeakAA = SingleLengthPSTs[i].AminoAcidSymbol;
//		if (HomePeak == HopPeak)
//		{
//			int insert_ptr = atomicAdd(&multipleLengthPstCounter, 1);
//			//int ins_ptr = atomicAdd(&charArrayCounter, 1);
//			IndexToInsertAA = IndexToInsertAA + 1;
//			char *tempPstArrayCopy;
//			my_strcpy(tempPstArrayCopy, tempPstArray);
//			tempPstArrayCopy[IndexToInsertAA] = HopPeakAA;
//			//(MultipleLengthPst_ptr[insert_ptr].PstTag) = tempPstArray;
//			my_strcpy(MultipleLengthPst_ptr[insert_ptr].PstTag, tempPstArrayCopy);
//			//MultipleLengthPst_ptr[insert_ptr].PstTag[0] = HomePeakAA;
//			//MultipleLengthPst_ptr[insert_ptr].PstTag[1] = HopPeakAA;
//			//GeneratingMultipleLengthPstsInner(HopPeak, HopPeakAA, MultipleLengthPst_ptr, SingleLengthPSTs, tempPstArrayCopy, IndexToInsertAA, N);
//		}
//		else
//		{
//			int a = 1;
//		}
//	}
//	if (i == N)
//	{
//		return;
//	}
//	return;
//	//return;
//}

__device__ int fact(int f)
{
	if (f == 0)
		return 1;
	else
		return f * fact(f - 1);
}


__global__ void GeneratingMultipleLengthPsts(_PeptideSequenceTags *MultipleLengthPst_ptr, _DataForPsts *SingleLengthPSTs, int N)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < N)
	{
		int HomePeak = SingleLengthPSTs[tid].endIndex;
		char HomePeakAA = SingleLengthPSTs[tid].AminoAcidSymbol;
		int StartIndex = SingleLengthPSTs[tid].startIndex;
		for (int i = 0; i < N; i++)
		{
			int HopPeak = SingleLengthPSTs[i].startIndex;
			char HopPeakAA = SingleLengthPSTs[i].AminoAcidSymbol;
			int EndIndex = SingleLengthPSTs[i].endIndex;
			if (HomePeak == HopPeak)
			{
				int insert_ptr = atomicAdd(&multipleLengthPstCounter, 1);
				MultipleLengthPst_ptr[insert_ptr].PstTag[0] = HomePeakAA;
				MultipleLengthPst_ptr[insert_ptr].PstTag[1] = HopPeakAA;
				MultipleLengthPst_ptr[insert_ptr].startIndex = StartIndex;
				MultipleLengthPst_ptr[insert_ptr].endIndex = EndIndex;
			}
		}
		//char tempPstArray[8];
		//int IndexToInsertAA = 0;
		//tempPstArray[IndexToInsertAA] = HomePeakAA;
		//int a = fact(5);
		//GeneratingMultipleLengthPstsInner(HomePeak, HomePeakAA, MultipleLengthPst_ptr, SingleLengthPSTs, tempPstArray, IndexToInsertAA, N);
		/*GeneratingMultipleLengthPstsInner(HomePeak, HomePeakAA);*/
	}
	/*for (int i = 0; i < 2; i++)
	{
		for (int j=0; j<3; j++)
			push_back_multipleLengthPsts(MultipleLengthPst_ptr, 'L', 'G');
	}*/
}

//__global__ void GeneratingMultipleLengthPsts2(_PeptideSequenceTags *MultipleLengthPst_ptr, _PeptideSequenceTags *DupletPSTs, _DataForPsts *SingleLengthPSTs, int N)
//{
//	int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	if (tid < N)
//	{
//		int HomePeak = DupletPSTs[tid].endIndex;
//		char HomePeakAA = DupletPSTs[tid].PstTag[0];
//		char HomePeakAA2 = DupletPSTs[tid].PstTag[1];
//		int size = DupletPSTs[tid].PstTag;
//		int StartIndex = DupletPSTs[tid].startIndex;
//		for (int i = 0; i < N; i++)
//		{
//			int HopPeak = SingleLengthPSTs[i].startIndex;
//			char HopPeakAA = SingleLengthPSTs[i].AminoAcidSymbol;
//			int EndIndex = SingleLengthPSTs[tid].endIndex;
//			if (HomePeak == HopPeak)
//			{
//				int insert_ptr = atomicAdd(&multipleLengthPstCounter, 1);
//				MultipleLengthPst_ptr[insert_ptr].PstTag[0] = HomePeakAA;
//				MultipleLengthPst_ptr[insert_ptr].PstTag[1] = HomePeakAA2;
//				MultipleLengthPst_ptr[insert_ptr].PstTag[2] = HopPeakAA;
//				MultipleLengthPst_ptr[insert_ptr].startIndex = StartIndex;
//				MultipleLengthPst_ptr[insert_ptr].endIndex = EndIndex;
//			}
//			else
//			{
//				int a = 0;
//			}
//		}
//	}
//}

__global__ void GeneratingMultipleLengthPsts2(_PeptideSequenceTags *MultipleLengthPst_ptr, _PeptideSequenceTags *DupletPSTs, _DataForPsts *SingleLengthPSTs, int N, int num, int SizeOfPst)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < N)
	{
		int HomePeak = DupletPSTs[tid].endIndex;
		char *HomePeakAA = DupletPSTs[tid].PstTag;
		int StartIndex = DupletPSTs[tid].startIndex;
		for (int i = 0; i < num; i++)
		{
			int HopPeak = SingleLengthPSTs[i].startIndex;
			char HopPeakAA = SingleLengthPSTs[i].AminoAcidSymbol;
			int EndIndex = SingleLengthPSTs[i].endIndex;
			if (HomePeak == HopPeak)
			{
				int insert_ptr = atomicAdd(&multipleLengthPstCounter, 1);
				my_strcpy(MultipleLengthPst_ptr[insert_ptr].PstTag, HomePeakAA);
				MultipleLengthPst_ptr[insert_ptr].PstTag[SizeOfPst-1] = HopPeakAA;
				MultipleLengthPst_ptr[insert_ptr].startIndex = StartIndex;
				MultipleLengthPst_ptr[insert_ptr].endIndex = EndIndex;
			}
			else
			{
				int a = 0;
			}
		}
	}
}

string convertToString(char* a, int size)
{
	int i;
	string s = "";
	for (i = 0; i < size; i++) {
		s = s + a[i];
	}
	return s;
}

extern "C" __declspec(dllexport) double __cdecl
wholeproteinmasstuner(double PeakListMasses[], double PeakListIntensities[], int PeakListLength, double MwTolerance, double NeutralLoss, double Slider_Value, double HopThreshold)
{
	double WholeProteinMass = PeakListMasses[0];

	const int N = PeakListLength;
	const int zN = (floor(PeakListLength*PeakListLength) / 2) - (floor(PeakListLength / 2));
	double *dev_masses, *dev_intensities;

	double aminoAcidMassesList[21] = { 57.0214600000000, 71.0371100000000, 87.0320300000000, 97.0527600000000, 99.0684100000000, 101.047680000000, 103.009190000000, 113.084060000000, 114.042930000000, 115.026940000000, 128.058580000000, 128.094960000000, 129.042590000000, 131.040490000000, 137.058910000000, 147.068410000000, 156.101110000000, 163.063330000000, 168.964203000000, 186.079310000000, 255.158295000000 };
	char aminoAcidSymbolList[21] = { 'G', 'A', 'S', 'P', 'V', 'T', 'C', 'L', 'N', 'D', 'Q', 'K', 'E', 'M', 'H', 'F', 'R', 'Y', 'U', 'W', 'O' };
	
	double *dev_aminoAcidMassesList;
	char *dev_aminoAcidSymbolList;

	hipMalloc((void**)&dev_masses, sizeof(double) * N);
	hipMalloc((void**)&dev_intensities, sizeof(double) * N);
	hipMalloc((void**)&dev_aminoAcidMassesList, sizeof(double)*21);
	hipMalloc((void**)&dev_aminoAcidSymbolList, sizeof(char)*21);

	thrust::device_vector<double> DevicePeakListMassesSum(zN);
	double *DevicePeakListMassesSum_ptr = thrust::raw_pointer_cast(DevicePeakListMassesSum.data());

	thrust::device_vector<double> DevicePeakListAvgIntensities(zN);
	double *DevicePeakListAvgIntensities_ptr = thrust::raw_pointer_cast(DevicePeakListAvgIntensities.data());

	thrust::device_vector<_DataForPsts> SingleLengthPSTs(zN);
	_DataForPsts *SingleLengthPSTs_ptr = thrust::raw_pointer_cast(SingleLengthPSTs.data());

	hipMemcpy(dev_masses, PeakListMasses, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_intensities, PeakListIntensities, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_aminoAcidMassesList, aminoAcidMassesList, 21 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_aminoAcidSymbolList, aminoAcidSymbolList, 21 * sizeof(char), hipMemcpyHostToDevice);

	//int Count = 0;
	//int *devCount;
	//hipMalloc((void**)&devCount, sizeof(int));
	//hipMemcpy(devCount, &Count, sizeof(int), hipMemcpyHostToDevice);

	int THREADS = 256;
	int BLOCKS = (N + THREADS - 1);
	vectorAdd << < BLOCKS, THREADS >> > (dev_masses, dev_intensities, DevicePeakListMassesSum_ptr, DevicePeakListAvgIntensities_ptr, SingleLengthPSTs_ptr, dev_aminoAcidMassesList, dev_aminoAcidSymbolList, MwTolerance, NeutralLoss, HopThreshold, N);
	//hipMemcpy(&Count, devCount, sizeof(int), hipMemcpyHostToDevice);

	thrust::host_vector<_DataForPsts> Host_SingleLengthPSTs = SingleLengthPSTs;
	thrust::host_vector<double> PeakListMassesSum = DevicePeakListMassesSum;
	thrust::host_vector<double> PeakListIntensitiesAverage = DevicePeakListAvgIntensities;
	thrust::host_vector<_ShortlistedMassSumsAndIntensities> shortlistedMassSumAndIntensities;

	for (int i = 0; i < zN; i++)
	{
		double a = PeakListMassesSum[i];
		if (PeakListMasses[0] - MwTolerance <= PeakListMassesSum[i] && PeakListMassesSum[i] <= PeakListMasses[0] + MwTolerance)
		{
			_ShortlistedMassSumsAndIntensities data;
			data.massSum = PeakListMassesSum[i];
			data.AvgIntensity = PeakListIntensitiesAverage[i];
			shortlistedMassSumAndIntensities.push_back(data);
		}
	}

	std::sort(shortlistedMassSumAndIntensities.begin(), shortlistedMassSumAndIntensities.end(),
		[](const _ShortlistedMassSumsAndIntensities &mass, const _ShortlistedMassSumsAndIntensities &mass2)
	{ return (mass.massSum < mass2.massSum); });
	// from here
	thrust::device_vector<_DataForPsts> Final_SingleLengthPSTs;
	int startIndex[100];
	int endIndex[100];
	double startIndexMass[100];
	double endIndexMass[100];
	double massDifferenceBetweenPeaks[100];
	char AminoAcidSymbol[100];
	double TagError[100];
	double averageIntensity[100];
	int k = 0;
	for (int i = 0; i < zN; i++)
	{
		if (Host_SingleLengthPSTs[i].startIndexMass == 0)
		{
			k = i;
			break;
		}
		else
		{
			startIndex[i] = Host_SingleLengthPSTs[i].startIndex;
			endIndex[i] = Host_SingleLengthPSTs[i].endIndex;
			startIndexMass[i] = Host_SingleLengthPSTs[i].startIndexMass;
			endIndexMass[i] = Host_SingleLengthPSTs[i].endIndexMass;
			massDifferenceBetweenPeaks[i] = Host_SingleLengthPSTs[i].massDifferenceBetweenPeaks;
			AminoAcidSymbol[i] = Host_SingleLengthPSTs[i].AminoAcidSymbol;
			TagError[i] = Host_SingleLengthPSTs[i].TagError;
			averageIntensity[i] = Host_SingleLengthPSTs[i].averageIntensity;
			Final_SingleLengthPSTs.push_back(Host_SingleLengthPSTs[i]);
		}
	}
	k = k;
	startIndex[0] = startIndex[0];
	endIndex[0] = endIndex[0];
	startIndexMass[0] = startIndexMass[0];
	endIndexMass[0] = endIndexMass[0];
	massDifferenceBetweenPeaks[0] = massDifferenceBetweenPeaks[0];
	AminoAcidSymbol[0] = AminoAcidSymbol[0];
	TagError[0] = TagError[0];
	averageIntensity[0] = averageIntensity[0];

	Final_SingleLengthPSTs[1] = Final_SingleLengthPSTs[1];
	_DataForPsts *Final_SingleLengthPSTs_ptr = thrust::raw_pointer_cast(Final_SingleLengthPSTs.data());

	/*thrust::device_vector<_PeptideSequenceTags> dev_MultipleLengthPst(Final_SingleLengthPSTs.size());*/
	thrust::device_vector<_PeptideSequenceTags> dev_MultipleLengthPst(1000);
	_PeptideSequenceTags *MultipleLengthPst_ptr = thrust::raw_pointer_cast(dev_MultipleLengthPst.data());

	int THREADS3 = 256;
	int BLOCKS3 = N + THREADS3 - 1;;
	int numOfThr = Final_SingleLengthPSTs.size();
	GeneratingMultipleLengthPsts << <BLOCKS3, THREADS3 >> > (MultipleLengthPst_ptr, Final_SingleLengthPSTs_ptr, numOfThr);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
	thrust::host_vector<_PeptideSequenceTags> host_MultipleLengthPst = dev_MultipleLengthPst;
	thrust::device_vector<_PeptideSequenceTags> dev_MultipleLengthPst2;
	
	int b = 0;
	for (int i = 0; i < host_MultipleLengthPst.size(); i++)
	{
		if (host_MultipleLengthPst[i].endIndex == 0)
		{
			b = i;
			break;
		}	
		dev_MultipleLengthPst2.push_back(host_MultipleLengthPst[i]);
		char *array = host_MultipleLengthPst[i].PstTag;
		int startInd = host_MultipleLengthPst[i].startIndex;
		int endInd = host_MultipleLengthPst[i].endIndex;
	}
	b = b;
	_PeptideSequenceTags *MultipleLengthPst_ptr2 = thrust::raw_pointer_cast(dev_MultipleLengthPst2.data());
	numOfThr = dev_MultipleLengthPst2.size();
	int PstSize = 3;
	GeneratingMultipleLengthPsts2 << <BLOCKS3, THREADS3 >> > (MultipleLengthPst_ptr, MultipleLengthPst_ptr2, Final_SingleLengthPSTs_ptr, numOfThr, Final_SingleLengthPSTs.size(), PstSize);
	host_MultipleLengthPst = dev_MultipleLengthPst;
	thrust::host_vector<_PeptideSequenceTags> host_MultipleLengthPst2 = dev_MultipleLengthPst2;
	int size = host_MultipleLengthPst2.size();
	dev_MultipleLengthPst2.clear();
	int h = 0;
	vector<char*> array;
	int StartIndArray[116];
	int EndIndArray[116];

	for (int i = size; i < host_MultipleLengthPst.size(); i++)
	{
		if (host_MultipleLengthPst[i].endIndex == 0)
		{
			h = i;
			break;
		}
		array.push_back(host_MultipleLengthPst[i].PstTag);
		int gfsd = host_MultipleLengthPst[i].startIndex;
		int hshfla = host_MultipleLengthPst[i].endIndex;
		StartIndArray[i] = host_MultipleLengthPst[i].startIndex;
		EndIndArray[i] = host_MultipleLengthPst[i].endIndex;
		dev_MultipleLengthPst2.push_back(host_MultipleLengthPst[i]);
	}
	StartIndArray[0] = StartIndArray[0];
	EndIndArray[0] = EndIndArray[0];
	//array[0] = array[0];
	h = h;
	PstSize = 4;
	numOfThr = dev_MultipleLengthPst2.size();
	GeneratingMultipleLengthPsts2 << <BLOCKS3, THREADS3 >> > (MultipleLengthPst_ptr, MultipleLengthPst_ptr2, Final_SingleLengthPSTs_ptr, numOfThr, Final_SingleLengthPSTs.size(), PstSize);
	host_MultipleLengthPst = dev_MultipleLengthPst;
	host_MultipleLengthPst2 = dev_MultipleLengthPst2;
	size = size + host_MultipleLengthPst2.size();
	dev_MultipleLengthPst2.clear();
	array.clear();
	for (int i = size; i < host_MultipleLengthPst.size(); i++)
	{
		if (host_MultipleLengthPst[i].endIndex == 0)
		{
			h = i;
			break;
		}
		array.push_back(host_MultipleLengthPst[i].PstTag);
		dev_MultipleLengthPst2.push_back(host_MultipleLengthPst[i]);
	}
	//array[0] = array[0];
	h = h;
	PstSize = 5;
	numOfThr = dev_MultipleLengthPst2.size();
	GeneratingMultipleLengthPsts2 << <BLOCKS3, THREADS3 >> > (MultipleLengthPst_ptr, MultipleLengthPst_ptr2, Final_SingleLengthPSTs_ptr, numOfThr, Final_SingleLengthPSTs.size(), PstSize);
	host_MultipleLengthPst = dev_MultipleLengthPst;
	host_MultipleLengthPst2 = dev_MultipleLengthPst2;
	size = size + host_MultipleLengthPst2.size();
	dev_MultipleLengthPst2.clear();
	array.clear();
	string arr[1000];
	for (int i = size; i < host_MultipleLengthPst.size(); i++)
	{
		if (host_MultipleLengthPst[i].endIndex == 0)
		{
			h = i;
			break;
		}
		arr[i- size] = convertToString(host_MultipleLengthPst[i].PstTag, 8);
		dev_MultipleLengthPst2.push_back(host_MultipleLengthPst[i]);
	}
	arr[0] = arr[0];
	//array[0] = array[0];
	h = h;
	PstSize = 6;
	numOfThr = dev_MultipleLengthPst2.size();
	GeneratingMultipleLengthPsts2 << <BLOCKS3, THREADS3 >> > (MultipleLengthPst_ptr, MultipleLengthPst_ptr2, Final_SingleLengthPSTs_ptr, numOfThr, Final_SingleLengthPSTs.size(), PstSize);
	host_MultipleLengthPst = dev_MultipleLengthPst;
	host_MultipleLengthPst2 = dev_MultipleLengthPst2;
	size = size + host_MultipleLengthPst2.size();
	dev_MultipleLengthPst2.clear();
	string arr2[1000];
	for (int i = size; i < host_MultipleLengthPst.size(); i++)
	{
		if (host_MultipleLengthPst[i].endIndex == 0)
		{
			h = i;
			break;
		}
		arr2[i- size] = convertToString(host_MultipleLengthPst[i].PstTag, 8);
		dev_MultipleLengthPst2.push_back(host_MultipleLengthPst[i]);
	}
	arr2[0] = arr2[0];
	//array[0] = array[0];
	h = h;
	PstSize = 7;
	numOfThr = dev_MultipleLengthPst2.size();
	GeneratingMultipleLengthPsts2 << <BLOCKS3, THREADS3 >> > (MultipleLengthPst_ptr, MultipleLengthPst_ptr2, Final_SingleLengthPSTs_ptr, numOfThr, Final_SingleLengthPSTs.size(), PstSize);
	host_MultipleLengthPst = dev_MultipleLengthPst;
	host_MultipleLengthPst2 = dev_MultipleLengthPst2;
	size = size + host_MultipleLengthPst2.size();
	dev_MultipleLengthPst2.clear();

	for (int i = size; i < host_MultipleLengthPst.size(); i++)
	{
		if (host_MultipleLengthPst[i].endIndex == 0)
		{
			h = i;
			break;
		}
		arr[i- size] = convertToString(host_MultipleLengthPst[i].PstTag, 8);
		dev_MultipleLengthPst2.push_back(host_MultipleLengthPst[i]);
	}
	arr[0] = arr[0];
	StartIndArray[0] = StartIndArray[0];
	EndIndArray[0] = EndIndArray[0];
	//array[0] = array[0];
	h = h;
	PstSize = 8;
	numOfThr = dev_MultipleLengthPst2.size();
	GeneratingMultipleLengthPsts2 << <BLOCKS3, THREADS3 >> > (MultipleLengthPst_ptr, MultipleLengthPst_ptr2, Final_SingleLengthPSTs_ptr, numOfThr, Final_SingleLengthPSTs.size(), PstSize);
	// till here
	double minSum = shortlistedMassSumAndIntensities[0].massSum;
	double maxSum = shortlistedMassSumAndIntensities[shortlistedMassSumAndIntensities.size() - 1].massSum;

	int sizeOfShortlistedData = shortlistedMassSumAndIntensities.size();
	double SliderValue = (WholeProteinMass * Slider_Value) / (pow(10.0, 6.0));
	int NumOfThreadsToLaunch = floor((maxSum - minSum) * (1 / SliderValue));

	thrust::device_vector<_ShortlistedMassSumsAndIntensities> device_shortlistedMassSumAndIntensities = shortlistedMassSumAndIntensities;
	_ShortlistedMassSumsAndIntensities *raw_ptr = thrust::raw_pointer_cast(device_shortlistedMassSumAndIntensities.data());

	thrust::device_vector<_WindowCapturedElementsStruct> device_windowcapturedelements(NumOfThreadsToLaunch);
	_WindowCapturedElementsStruct *raw_ptr2 = thrust::raw_pointer_cast(device_windowcapturedelements.data());

	int THREADS2 = 256;
	int BLOCKS2 = (NumOfThreadsToLaunch / THREADS + 1);

	//WindowLaunchKernel << <BLOCKS2, THREADS2 >> > (NumOfThreadsToLaunch, minSum, maxSum, raw_ptr, sizeOfShortlistedData, raw_ptr2, SliderValue);

	//thrust::host_vector<_WindowCapturedElementsStruct> host_windowcapturedelements = device_windowcapturedelements;

	//double TunedMass = 0;
	//int oldElementCount = 0;

	//for (int x = 0; x < NumOfThreadsToLaunch; x++)
	//{
	//	if (oldElementCount < host_windowcapturedelements[x].elementCount)
	//	{
	//		oldElementCount = host_windowcapturedelements[x].elementCount;
	//		TunedMass = host_windowcapturedelements[x].TunedMass;
	//	}
	//	else if (oldElementCount == host_windowcapturedelements[x].elementCount)
	//	{
	//		if (abs(TunedMass - WholeProteinMass) >= abs(host_windowcapturedelements[x].TunedMass - WholeProteinMass))
	//		{
	//			TunedMass = host_windowcapturedelements[x].TunedMass;
	//		}
	//	}
	//}

	
	hipDeviceSynchronize();
	return 1.1;
}
